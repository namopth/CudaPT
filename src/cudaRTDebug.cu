#include "hip/hip_runtime.h"
#include "cudaRTCommon.h"

#define BLOCK_SIZE 16

namespace cudaRTDebug
{

	float* g_devResultData = nullptr;
	size_t g_resultDataSize = 0;

	struct ShootRayResult
	{
		float4 light;
	};

	__device__ ShootRayResult ptDebug_normalRay(const CURay& ray, RTVertex* vertices, RTTriangle* triangles, RTMaterial* materials, CURTTexture* textures)
	{
		ShootRayResult rayResult;

		TracePrimitiveResult traceResult;
		if (TracePrimitive(ray, traceResult))
		{
			RTTriangle* tri = &triangles[traceResult.triId];
			RTMaterial* mat = &materials[tri->matInd];
			RTVertex* v0 = &vertices[tri->vertInd0];
			RTVertex* v1 = &vertices[tri->vertInd1];
			RTVertex* v2 = &vertices[tri->vertInd2];
			float2 uv0 = make_float2(v0->tex._x, v0->tex._y);
			float2 uv1 = make_float2(v1->tex._x, v1->tex._y);
			float2 uv2 = make_float2(v2->tex._x, v2->tex._y);
			float2 uv = uv0 * traceResult.w + uv1 * traceResult.u + uv2 * traceResult.v;
			float3 n0 = V32F3(v0->norm);
			float3 n1 = V32F3(v1->norm);
			float3 n2 = V32F3(v2->norm);
			float3 norm = n0 * traceResult.w + n1 * traceResult.u + n2 * traceResult.v;

			float4 diff;
			float3 ambient;
			float3 specular;
			float3 emissive;
			GetMaterialColors(mat, uv, textures, diff, ambient, specular, emissive);

			float3 w = norm;
			float3 u = normalize(vecCross((fabs(w.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), w));
			float3 v = vecCross(w, u);

			rayResult.light = diff;
		}
		else
		{
			rayResult.light.x = 1.f;
			rayResult.light.y = 1.f;
			rayResult.light.z = 1.f;
			rayResult.light.w = 1.f;
		}

		return rayResult;
	}

	uint32 WangHash(uint32 a) {
		a = (a ^ 61) ^ (a >> 16);
		a = a + (a << 3);
		a = a ^ (a >> 4);
		a = a * 0x27d4eb2d;
		a = a ^ (a >> 15);
		return a;
	}

	__global__ void ptDebug_kernel(float3 camPos, float3 camDir, float3 camUp, float3 camRight, float fov,
		float width, float height, RTVertex* vertices, RTTriangle* triangles, RTMaterial* materials, CURTTexture* textures
		, float* result)
	{
		uint x = blockIdx.x * blockDim.x + threadIdx.x;
		uint y = blockIdx.y * blockDim.y + threadIdx.y;
		uint ind = (y * width + x) * 3;
		int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;


		float u = (2.f * ((float)x + 0.5f) / width - 1.f) * tan(fov * 0.5f) * width / height;
		float v = (2.f * ((float)y + 0.5f) / height - 1.f) * tan(fov * 0.5f);
		float3 dir = normalize(camRight * u + camUp * v + camDir);
		CURay ray(camPos, dir);

		ShootRayResult rayResult = ptDebug_normalRay(ray, vertices, triangles, materials, textures);

		result[ind] = rayResult.light.x;
		result[ind + 1] = rayResult.light.y;
		result[ind + 2] = rayResult.light.z;
	}

	void cudaDebugClean()
	{
		freeAllBVHCudaMem();
		CUFREE(g_devResultData);
	}

	bool cudaDebugRender(NPMathHelper::Vec3 camPos, NPMathHelper::Vec3 camDir, NPMathHelper::Vec3 camUp, float fov, RTScene* scene
		, float width, float height, float* result)
	{
		// Check and allocate everything
		if (!scene || !scene->GetCompactBVH()->IsValid())
			return false;

		NPMathHelper::Vec3 camRight = camDir.cross(camUp).normalize();
		camUp = camRight.cross(camDir).normalize();

		if (!g_bIsCudaInit || scene->GetIsCudaDirty())
		{
			initAllBVHCudaMem(scene);
		}

		if (!g_bIsCudaInit)
			return false;

		if (!g_devResultData || g_resultDataSize != (sizeof(float) * 3 * width * height))
		{
			g_resultDataSize = sizeof(float) * 3 * width * height;
			CUFREE(g_devResultData);
			hipMalloc((void**)&g_devResultData, g_resultDataSize);
		}

		float3 f3CamPos = V32F3(camPos);
		float3 f3CamUp = V32F3(camUp);
		float3 f3CamDir = V32F3(camDir);
		float3 f3CamRight = V32F3(camRight);

		// Kernel go here
		dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
		dim3 grid(width / block.x, height / block.y, 1);
		ptDebug_kernel << < grid, block >> > (f3CamPos, f3CamDir, f3CamUp, f3CamRight, fov, width, height, g_devVertices, g_devTriangles, g_devMaterials, g_devTextures
			, g_devResultData);

		// Copy result to host
		hipMemcpy(result, g_devResultData, g_resultDataSize, hipMemcpyDeviceToHost);
		return true;
	}

};