#include "hip/hip_runtime.h"
#include "cudaRTCommon.h"

#define BLOCK_SIZE 16
#define NORMALRAY_BOUND_MAX 10
namespace cudaRTPTRegen
{
	CUDA_RT_COMMON_ATTRIBS_N(0)
	CUDA_RT_COMMON_ATTRIBS_BGN
	CUDA_RT_COMMON_ATTRIBS_END

	float* g_devResultData = nullptr;
	float* g_devAccResultData = nullptr;

	NPMathHelper::Mat4x4 g_matLastCamMat;
	NPMathHelper::Mat4x4 g_matCurCamMat;
	uint32 g_uCurFrameN;
	size_t g_resultDataSize = 0;

	struct TracerData
	{
		float2* winSize;
		float fov;
		float3* camOrig;
		float3* camU;
		float3* camR;
		float3* camD;
		RTVertex* vertices;
		RTTriangle* triangles;
		RTMaterial* materials;
		CURTTexture* textures;
		hiprandState *randstate;

		__device__ TracerData(float2* _winSize, float _fov,float3* _camOrig, float3* _camU, float3* _camR, float3* _camD
			, RTVertex* _vertices, RTTriangle* _triangles, RTMaterial* _materials, CURTTexture* _textures, hiprandState* _randState)
			: winSize(_winSize)
			, fov(_fov)
			, camOrig(_camOrig)
			, camU(_camU)
			, camR(_camR)
			, camD(_camD)
			, vertices(_vertices)
			, triangles(_triangles)
			, materials(_materials)
			, textures(_textures)
			, randstate(_randState)
		{}
	};

	struct PTSample
	{
		uint2 pixel;
		float2 uv;
		float pixelContrib;
		uint pathDepth;
		uint sampleTime;
		float3 sampleResult;
		float3 accumResult;

		__device__ PTSample(uint2 _pixel, float2 _uv)
			: pixel(_pixel), uv(_uv), pixelContrib(1.0f), pathDepth(0), sampleTime(0), accumResult()
		{
			accumResult = sampleResult = make_float3(0.f, 0.f, 0.f);
		}
	};

	enum RAYTYPE
	{
		RAYTYPE_EYE = 0,
		RAYTYPE_DIFF = 1,
		RAYTYPE_SPEC = 2
	};

	template <int depth = 0>
	__device__ void pt0_normalRay(const CURay& ray, RAYTYPE rayType, PTSample& sample, TracerData& tracerData)
	{
		TracePrimitiveResult traceResult;
		if (TracePrimitive(ray, traceResult, M_INF, M_FLT_BIAS_EPSILON, false))
		{
			RTTriangle* tri = &tracerData.triangles[traceResult.triId];
			RTMaterial* mat = &tracerData.materials[tri->matInd];
			RTVertex* v0 = &tracerData.vertices[tri->vertInd0];
			RTVertex* v1 = &tracerData.vertices[tri->vertInd1];
			RTVertex* v2 = &tracerData.vertices[tri->vertInd2];
			float2 uv0 = make_float2(v0->tex._x, v0->tex._y);
			float2 uv1 = make_float2(v1->tex._x, v1->tex._y);
			float2 uv2 = make_float2(v2->tex._x, v2->tex._y);
			float2 uv = uv0 * traceResult.w + uv1 * traceResult.u + uv2 * traceResult.v;
			float3 n0 = V32F3(v0->norm);
			float3 n1 = V32F3(v1->norm);
			float3 n2 = V32F3(v2->norm);
			float3 norm = n0 * traceResult.w + n1 * traceResult.u + n2 * traceResult.v;

			float3 diff;
			float3 emissive;
			float trans;
			float specular;
			float metallic;
			float roughness;
			float anisotropic;
			float sheen;
			float sheenTint;
			float clearcoat;
			float clearcoatGloss;
			GetMaterialColors(mat, uv, tracerData.textures, diff, norm, emissive, trans, specular, metallic, roughness
				, anisotropic, sheen, sheenTint, clearcoat, clearcoatGloss);
			float3 nl = vecDot(norm, ray.dir) < 0.f ? norm : -1 * norm;
			{
				// Get some random microfacet
				float3 hDir = ImportanceSampleGGX(make_float2(hiprand_uniform(tracerData.randstate), hiprand_uniform(tracerData.randstate)), roughness, nl);

				// Calculate flesnel
				float voH = vecDot(-1 * ray.dir, hDir);
				float3 f0 = vecLerp(0.08 * make_float3(specular, specular, specular), diff, metallic);
				float3 brdf_f = Fresnel(f0, voH);

				// Reflected or Refracted
				float reflProb = lerp(length(brdf_f), 1.0f, metallic);
				float refrProb = trans;
				float3 reflDir;
				float3 refrDir;

				CURay nextRay = ray;
				float3 lightMulTerm;
				RAYTYPE nextRayType = rayType;

				if (refrProb > 0)
				{
					bool into = vecDot(nl, norm) > 0.f;
					float nt = specular * 0.8f + 1.f;
					float nc = 1.0f;
					float nnt = into ? nc / nt : nt / nc;
					float ddn = vecDot(hDir, ray.dir);
					float cos2t = 1.f - nnt * nnt *(1.f - ddn * ddn);
					if (cos2t < 0.f)
					{
						reflProb = 1.0f;// refrProb = 0.f;
					}
					else
					{
						refrDir = normalize(ray.dir * nnt - hDir * (ddn*nnt + sqrtf(cos2t)));
					}
				}

				if (reflProb > 0)
				{
					reflDir = normalize(ray.dir - hDir * 2 * vecDot(hDir,ray.dir));
					if (vecDot(reflDir, nl) < 0.f)
						reflProb = 0.f;
				}

				// Reflected
				if (ProbabilityRand(tracerData.randstate, reflProb))
				{
					nextRay = CURay(ray.orig + (traceResult.dist - M_FLT_BIAS_EPSILON) * ray.dir, reflDir);
					// ShootRayResult nextRayResult = pt0_normalRay<depth + 1>(nextRay, vertices, triangles, materials, textures, randstate);

					// Microfacet specular = D*G*F / (4*NoL*NoV)
					// pdf = D * NoH / (4 * VoH)
					// (G * F * VoH) / (NoV * NoH)
					float VoH = vecDot(-1 * ray.dir, hDir);
					float NoV = vecDot(nl, -1 * ray.dir);
					float NoH = vecDot(nl, hDir);
					float NoL = vecDot(nl, reflDir);
					float G = GeometricVisibility(roughness, NoV, NoL, VoH);
					//shadeResult = vecMul((brdf_f * G * VoH) / (NoV * NoH * reflProb) , nextRayResult.light) + emissive;
					lightMulTerm = (brdf_f * G * VoH) / (NoV * NoH * reflProb);
					nextRayType = RAYTYPE_SPEC;
				}

				// Diffused or Transmited
				else
				{
					// Transmited
					if (ProbabilityRand(tracerData.randstate, refrProb))
					{
						nextRay = CURay(ray.orig + (traceResult.dist + M_FLT_BIAS_EPSILON) * ray.dir + refrDir * M_FLT_BIAS_EPSILON, refrDir);
						//ShootRayResult nextRayResult = pt0_normalRay<depth + 1>(nextRay, vertices, triangles, materials, textures, randstate);
						float cosine = vecDot(-1 * nl, refrDir);
						//shadeResult = (cosine * vecMul(diff, nextRayResult.light)) / (refrProb * (1 - reflProb)) + emissive;
						lightMulTerm = cosine * diff / (refrProb * (1 - reflProb));
						nextRayType = RAYTYPE_SPEC;
					}
					// Diffused
					else
					{
						float3 w = nl;
						float3 u = normalize(vecCross((fabs(w.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), w));
						float3 v = vecCross(w, u);
						u = vecCross(v, w);

						float r1 = 2.f * M_PI * hiprand_uniform(tracerData.randstate);
						float r2cos = sqrtf(hiprand_uniform(tracerData.randstate));
						float r2sin = 1.f - r2cos*r2cos;
						float3 diffDir = normalize(w * r2cos + u * r2sin * cosf(r1) + v * r2sin * sinf(r1));

						nextRay = CURay(ray.orig + traceResult.dist * ray.dir + diffDir * M_FLT_BIAS_EPSILON, diffDir);
						//ShootRayResult nextRayResult = pt0_normalRay<depth + 1>(nextRay, vertices, triangles, materials, textures, randstate);

						float VoH = vecDot(-1 * ray.dir, hDir);
						float NoV = vecDot(nl, -1 * ray.dir);
						float NoL = vecDot(nl, diffDir);
						//shadeResult = (M_PI * vecMul(Diffuse(diff, roughness, NoV, NoL, VoH), nextRayResult.light)) / ((1 - refrProb) * (1 - reflProb)) + emissive;
						lightMulTerm = M_PI * Diffuse(diff, roughness, NoV, NoL, VoH)/ ((1 - refrProb) * (1 - reflProb));
						nextRayType = RAYTYPE_DIFF;
					}
				}

				sample.pixelContrib = sample.pixelContrib * length(lightMulTerm);
				float nextRayResult;

				if ((rayType == RAYTYPE_DIFF && nextRayType == RAYTYPE_SPEC) || length(emissive) > 0.f)
					sample.pixelContrib = 0.f;

				bool isAccum = (sample.pathDepth == 0);
				if (hiprand_uniform(tracerData.randstate) >= sample.pixelContrib)
				{
					sample.pixelContrib = 1.0f;
					sample.pathDepth = 0;
					sample.sampleTime++;
					nextRayType = RAYTYPE_EYE;
					float au = sample.uv.x + (hiprand_uniform(tracerData.randstate) - 0.5f) / tracerData.winSize->y * tan(tracerData.fov * 0.5f);
					float av = sample.uv.y + (hiprand_uniform(tracerData.randstate) - 0.5f) / tracerData.winSize->y * tan(tracerData.fov * 0.5f);
					float3 dir = normalize(*tracerData.camR * au + *tracerData.camU * av + *tracerData.camD);
					nextRay = CURay(*tracerData.camOrig, dir);
				}
				else
				{
					sample.pathDepth++;
				}

				pt0_normalRay<depth + 1>(nextRay, nextRayType, sample, tracerData);

				sample.sampleResult = vecMul(lightMulTerm, sample.sampleResult) + emissive;

				if (isAccum && sample.sampleTime > 0)
				{
					sample.accumResult = sample.accumResult + sample.sampleResult / (float)sample.sampleTime;
					sample.sampleResult = make_float3(0.f, 0.f, 0.f);
				}
			}
		}
		else
		{
			//sample.sampleTime++;
			sample.sampleResult = make_float3(0.f, 0.f, 0.f);
		}
	}

	template <>
	__device__ void pt0_normalRay<NORMALRAY_BOUND_MAX>(const CURay& ray, RAYTYPE rayType, PTSample& sample, TracerData& tracerData)
	{
		//sample.sampleTime++;
		sample.sampleResult = make_float3(0.f, 0.f, 0.f);
	}

	uint32 WangHash(uint32 a) {
		a = (a ^ 61) ^ (a >> 16);
		a = a + (a << 3);
		a = a ^ (a >> 4);
		a = a * 0x27d4eb2d;
		a = a ^ (a >> 15);
		return a;
	}

	__global__ void pt0_kernel(float3 camPos, float3 camDir, float3 camUp, float3 camRight, float fov,
		float width, float height, RTVertex* vertices, RTTriangle* triangles, RTMaterial* materials, CURTTexture* textures
		, uint32 frameN, uint32 hashedFrameN, float* result, float* accResult)
	{
		uint x = blockIdx.x * blockDim.x + threadIdx.x;
		uint y = blockIdx.y * blockDim.y + threadIdx.y;
		if (x >= width || y >= height)
			return;
		uint ind = (y * width + x) * 3;
		int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;


		float u = (2.f * ((float)x + 0.5f) / width - 1.f) * tan(fov * 0.5f) * width / height;
		float v = (2.f * ((float)y + 0.5f) / height - 1.f) * tan(fov * 0.5f);

		hiprandState randstate;
		hiprand_init(hashedFrameN + ind, 0, 0, &randstate);
		float au = u + (hiprand_uniform(&randstate) - 0.5f) / height * tan(fov * 0.5f);
		float av = v + (hiprand_uniform(&randstate) - 0.5f) / height * tan(fov * 0.5f);

		float3 dir = normalize(camRight * au + camUp * av + camDir);
		CURay ray(camPos, dir);

		PTSample sampleResult(make_uint2(x,y), make_float2(u, v));

		float2 winSize = make_float2(width, height);
		TracerData tracerData(&winSize, fov, &camPos, &camUp, &camRight, &camDir, vertices, triangles, materials, textures, &randstate);
		pt0_normalRay(ray, RAYTYPE_EYE, sampleResult, tracerData);

		float resultInf = 1.f / (float)(frameN + 1);
		float oldInf = 1.f - resultInf;
		result[ind] = max(resultInf * sampleResult.accumResult.x + oldInf * result[ind], 0.f);
		result[ind + 1] = max(resultInf * sampleResult.accumResult.y + oldInf * result[ind + 1], 0.f);
		result[ind + 2] = max(resultInf * sampleResult.accumResult.z + oldInf * result[ind + 2], 0.f);
	}

	void CleanMem()
	{
		freeAllBVHCudaMem();
		CUFREE(g_devResultData);
		CUFREE(g_devAccResultData);
	}

	bool Render(NPMathHelper::Vec3 camPos, NPMathHelper::Vec3 camDir, NPMathHelper::Vec3 camUp, float fov, RTScene* scene
		, float width, float height, float* result)
	{
		// Check and allocate everything
		if (!scene || !scene->GetCompactBVH()->IsValid())
			return false;

		NPMathHelper::Vec3 camRight = camDir.cross(camUp).normalize();
		camUp = camRight.cross(camDir).normalize();

		g_matLastCamMat = g_matCurCamMat;
		g_matCurCamMat = NPMathHelper::Mat4x4::lookAt(camPos, camPos + camDir, camUp);
		g_uCurFrameN = (g_matLastCamMat != g_matCurCamMat) ? 0 : g_uCurFrameN + 1;

		if (!g_bIsCudaInit || scene->GetIsCudaDirty())
		{
			g_matLastCamMat = g_matCurCamMat;
			g_uCurFrameN = 0;
			initAllSceneCudaMem(scene);
		}
		else if (scene->GetIsCudaMaterialDirty())
		{
			updateAllSceneMaterialsCudaMem(scene);
			g_uCurFrameN = 0;
		}

		if (!g_bIsCudaInit)
			return false;

		if (!g_devResultData || !g_devAccResultData || g_resultDataSize != (sizeof(float) * 3 * width * height))
		{
			g_resultDataSize = sizeof(float) * 3 * width * height;
			CUFREE(g_devResultData);
			hipMalloc((void**)&g_devResultData, g_resultDataSize);
			CUFREE(g_devAccResultData);
			hipMalloc((void**)&g_devAccResultData, g_resultDataSize);
		}

		float3 f3CamPos = V32F3(camPos);
		float3 f3CamUp = V32F3(camUp);
		float3 f3CamDir = V32F3(camDir);
		float3 f3CamRight = V32F3(camRight);

		// Kernel go here
		dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
		dim3 grid(ceil(width / (float)block.x), ceil(height / (float)block.y), 1);
		pt0_kernel << < grid, block >> > (f3CamPos, f3CamDir, f3CamUp, f3CamRight, fov, width, height, g_devVertices, g_devTriangles, g_devMaterials, g_devTextures
			, g_uCurFrameN, WangHash(g_uCurFrameN), g_devResultData, g_devAccResultData);

		// Copy result to host
		hipMemcpy(result, g_devResultData, g_resultDataSize, hipMemcpyDeviceToHost);
		return true;
	}
}