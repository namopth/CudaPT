#include "hip/hip_runtime.h"
#include "cudaRTCommon.h"

#include <thrust/remove.h>
#include <thrust/execution_policy.h>

#define BLOCK_SIZE 16
#define NORMALRAY_BOUND_MAX 5
#define PATHSTREAM_SIZE 1E4*64

#define LIGHTRAY_BOUND_MAX 5
#define LIGHTVERTEX_N 640

namespace cudaRTBDPTStream
{
	CUDA_RT_COMMON_ATTRIBS_N(0)
	CUDA_RT_COMMON_ATTRIBS_BGN
	CUDA_RT_COMMON_ATTRIBS_END

	struct LightVertex
	{
		float3 pos;
		float3 norm;
		float3 irrad;
		float3 irradDir;

		float3 diff;
		float3 emissive;
		float specular;
		float metallic;
		float roughness;

		__hd__ LightVertex()
		{
			pos = norm = irrad = irradDir = make_float3(0.f, 0.f, 0.f);
		}
	};

	LightVertex* g_devLightVertices = nullptr;
	uint g_uLightVerticesSize = 0;
	uint* g_devLightTri = nullptr;
	uint g_lightTriN = 0;

	void freeLightPathMem()
	{
		g_uLightVerticesSize = 0;
		g_lightTriN = 0;
		CUFREE(g_devLightVertices);
		CUFREE(g_devLightTri);
	}

	void allocateLightPathMem()
	{
		HANDLE_ERROR(hipMalloc((void**)&g_devLightVertices, sizeof(LightVertex) * LIGHTVERTEX_N));
		HANDLE_ERROR(hipMemset((void*)g_devLightVertices, 0, sizeof(LightVertex) * LIGHTVERTEX_N));
	}

void updateLightTriCudaMem(RTScene* scene)
{
	g_lightTriN = 0;
	CUFREE(g_devLightTri);
	std::vector<uint> lightTri;
	for (uint i = 0; i < scene->m_pTriangles.size(); i++)
	{
		if (NPMathHelper::Vec3::length(scene->m_pMaterials[scene->m_pTriangles[i].matInd].emissive) > 0.f)
			lightTri.push_back(i);
	}
	uint* tempLightTri = new uint[lightTri.size()];
	for (uint i = 0; i < lightTri.size(); i++)
	{
		tempLightTri[i] = lightTri[i];
	}
	g_lightTriN = lightTri.size();
	HANDLE_ERROR(hipMalloc((void**)&g_devLightTri, sizeof(uint) * g_lightTriN));
	HANDLE_ERROR(hipMemcpy(g_devLightTri, tempLightTri, sizeof(uint) * g_lightTriN, hipMemcpyHostToDevice));

	DEL_ARRAY(tempLightTri);
}

	enum RAYTYPE
	{
		RAYTYPE_EYE = 0,
		RAYTYPE_DIFF = 1,
		RAYTYPE_SPEC = 2,
		RAYTYPE_LIGHT = 3
	};

	struct PTPathVertex
	{
		uint isTerminated;
		uint2 pathPixel;
		float3 pathOutDir;
		float3 pathVertexPos;
		float3 pathOutMulTerm;
		RAYTYPE pathType;
		float3 pathSample;
		float3 pathAccumSample;
		uint pathSampleN;
		uint pathSampleDepth;
		hiprandState randState;

		// for connecting light path
		float3 pathInMulTerm;
		float3 pathInDir;
		float3 origNorm;
		float3 origDiff;
		float origMetallic;
		float origRoughness;
		float origSpecular;
		float origTrans;

		__device__ PTPathVertex(uint _isTerminated, uint2 _pathPixel, float3 _pathOutDir, float3 _pathVertexPos, RAYTYPE _pathType, hiprandState _randState)
			: isTerminated(_isTerminated)
			, pathPixel(_pathPixel)
			, pathOutDir(_pathOutDir)
			, pathVertexPos(_pathVertexPos)
			, pathOutMulTerm(make_float3(1.f, 1.f, 1.f))
			, pathType(_pathType)
			, pathSample(make_float3(0.f, 0.f, 0.f))
			, pathAccumSample(make_float3(0.f, 0.f, 0.f))
			, pathSampleN(0)
			, pathSampleDepth(0)
			, randState(_randState)
			, pathInMulTerm(make_float3(0.f, 0.f, 0.f))
			, pathInDir(make_float3(0.f, 0.f, 0.f))
			, origNorm(make_float3(0.f, 1.f, 0.f))
			, origDiff(make_float3(0.f, 0.f, 0.f))
			, origMetallic(0.f)
			, origRoughness(0.f)
			, origSpecular(0.f)
			, origTrans(0.f)
		{}
	};

	PTPathVertex* g_devPathQueue = nullptr;
	uint g_uPathQueueCur = 0;
	uint g_uPathQueueSize = 0;
	PTPathVertex** g_devPathStream = nullptr;
	PTPathVertex** g_devEyeLightConPathStream = nullptr;
	uint g_uPathStreamSize = PATHSTREAM_SIZE;

	void freeStreamMem()
	{
		g_uPathQueueCur = g_uPathQueueSize = 0;
		CUFREE(g_devPathQueue);
		CUFREE(g_devPathStream);
		CUFREE(g_devEyeLightConPathStream);
	}

	void allocateStreamMem(uint queueSize = 480000)
	{
		g_uPathQueueSize = queueSize;
		HANDLE_ERROR(hipMalloc((void**)&g_devPathQueue, sizeof(PTPathVertex) * g_uPathQueueSize));
		HANDLE_ERROR(hipMemset((void*)g_devPathQueue, 0, sizeof(PTPathVertex) * g_uPathQueueSize));

		HANDLE_ERROR(hipMalloc((void**)&g_devPathStream, sizeof(PTPathVertex*) * g_uPathStreamSize));
		HANDLE_ERROR(hipMemset((void*)g_devPathStream, 0, sizeof(PTPathVertex*) * g_uPathStreamSize));

		HANDLE_ERROR(hipMalloc((void**)&g_devEyeLightConPathStream, sizeof(PTPathVertex*) * g_uPathStreamSize));
		HANDLE_ERROR(hipMemset((void*)g_devEyeLightConPathStream, 0, sizeof(PTPathVertex*) * g_uPathStreamSize));
	}

#pragma region SHADING_FUNC
	__device__ float3 Diffuse_Lambert(float3 DiffuseColor)
	{
		return DiffuseColor * (1 / M_PI);
	}

	__device__ float Vis_SmithJointApprox(float Roughness, float NoV, float NoL)
	{
		float a = Roughness * Roughness;
		float Vis_SmithV = NoL * (NoV * (1 - a) + a);
		float Vis_SmithL = NoV * (NoL * (1 - a) + a);
		return 0.5 * rcpf(Vis_SmithV + Vis_SmithL);
		//float k = (Roughness * Roughness) / 2.0f; // (Roughness + 1) * (Roughness + 1) / 8.f;
		//return (NoV / (NoV * (1 - k) + k))*(NoL / (NoL * (1 - k) + k));
	}

	__device__ float D_GGX(float Roughness, float NoH)
	{
		float m = Roughness * Roughness;
		float m2 = m*m;
		float d = (NoH * m2 - NoH) * NoH + 1;
		return m2 / (M_PI*d*d);
	}

	__device__ float3 F_Schlick(float3 SpecularColor, float VoH)
	{
		float Fc = pow(1 - VoH, 5);
		float firstTerm = saturate(50.0 * SpecularColor.z) * Fc;
		return make_float3(firstTerm, firstTerm, firstTerm) + (1 - Fc) * SpecularColor;
	}

	__device__ float3 ImportanceSampleGGX(float2 Xi, float Roughness, float3 N)
	{
		float a = Roughness * Roughness;
		float Phi = 2 * M_PI * Xi.x;
		float CosTheta = sqrt((1 - Xi.y) / (1 + (a*a - 1) * Xi.y));
		float SinTheta = sqrt(1 - CosTheta * CosTheta);
		float3 H;
		H.x = SinTheta * cos(Phi);
		H.y = SinTheta * sin(Phi);
		H.z = CosTheta;
		//float3 UpVector = abs(N.z) < 0.999 ? make_float3(0, 0, 1) : make_float3(1, 0, 0);
		//float3 TangentX = normalize(vecCross(UpVector, N));
		//float3 TangentY = vecCross(N, TangentX);

		float3 w = N;
		float3 u = normalize(vecCross((fabs(w.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), w));
		float3 v = vecCross(w, u);
		u = vecCross(v, w);

		// Tangent to world space
		return (u * H.x + v * H.y + w * H.z);
	}

	__device__ float3 Diffuse(float3 DiffuseColor, float Roughness, float NoV, float NoL, float VoH)
	{
		return Diffuse_Lambert(DiffuseColor);
	}

	__device__ float Distribution(float Roughness, float NoH)
	{
		return D_GGX(Roughness, NoH);
	}

	__device__ float GeometricVisibility(float Roughness, float NoV, float NoL, float VoH)
	{
		return Vis_SmithJointApprox(Roughness, NoV, NoL);
	}

	__device__ float3 Fresnel(float3 SpecularColor, float VoH)
	{
		return F_Schlick(SpecularColor, VoH);
	}

#pragma endregion SHADING_FUNC

	float* g_devResultData = nullptr;
	float* g_devAccResultData = nullptr;

	NPMathHelper::Mat4x4 g_matLastCamMat;
	NPMathHelper::Mat4x4 g_matCurCamMat;
	uint32 g_uCurFrameN = 0;
	size_t g_resultDataSize = 0;

	uint32 WangHash(uint32 a) {
		a = (a ^ 61) ^ (a >> 16);
		a = a + (a << 3);
		a = a ^ (a >> 4);
		a = a * 0x27d4eb2d;
		a = a ^ (a >> 15);
		return a;
	}

	__global__ void pt_traceLight_kernel(RTVertex* vertices, RTTriangle* triangles, RTMaterial* materials, CURTTexture* textures, PTPathVertex** pathStream, uint activePathStreamSize, LightVertex* lightVertices, uint curLightVerticesSize)
	{
		uint x = blockIdx.x * blockDim.x + threadIdx.x;
		if (x >= activePathStreamSize || pathStream[x]->isTerminated) return;
		PTPathVertex* procVertex = pathStream[x];
		CURay ray = CURay(procVertex->pathVertexPos, procVertex->pathOutDir);
		TracePrimitiveResult traceResult;
		if (TracePrimitive(ray, traceResult, M_INF, M_FLT_BIAS_EPSILON, false))
		{
			RTTriangle* tri = &triangles[traceResult.triId];
			RTMaterial* mat = &materials[tri->matInd];
			RTVertex* v0 = &vertices[tri->vertInd0];
			RTVertex* v1 = &vertices[tri->vertInd1];
			RTVertex* v2 = &vertices[tri->vertInd2];
			float2 uv0 = make_float2(v0->tex._x, v0->tex._y);
			float2 uv1 = make_float2(v1->tex._x, v1->tex._y);
			float2 uv2 = make_float2(v2->tex._x, v2->tex._y);
			float2 uv = uv0 * traceResult.w + uv1 * traceResult.u + uv2 * traceResult.v;
			float3 n0 = V32F3(v0->norm);
			float3 n1 = V32F3(v1->norm);
			float3 n2 = V32F3(v2->norm);
			float3 norm = n0 * traceResult.w + n1 * traceResult.u + n2 * traceResult.v;
			float3 triPos = V32F3(v0->pos) * traceResult.w + V32F3(v1->pos) * traceResult.u + V32F3(v2->pos) * traceResult.v;

			float3 diff;
			float3 emissive;
			float trans;
			float specular;
			float metallic;
			float roughness;
			float anisotropic;
			float sheen;
			float sheenTint;
			float clearcoat;
			float clearcoatGloss;
			GetMaterialColors(mat, uv, textures, diff, norm, emissive, trans, specular, metallic, roughness
				, anisotropic, sheen, sheenTint, clearcoat, clearcoatGloss);
			float3 nl = vecDot(norm, ray.dir) < 0.f ? norm : -1 * norm;

			lightVertices[curLightVerticesSize + x].irrad = procVertex->pathSample;
			lightVertices[curLightVerticesSize + x].irradDir = -1 * ray.dir;
			lightVertices[curLightVerticesSize + x].norm = nl;
			lightVertices[curLightVerticesSize + x].pos = triPos;
			lightVertices[curLightVerticesSize + x].diff = diff;
			lightVertices[curLightVerticesSize + x].emissive = emissive;
			lightVertices[curLightVerticesSize + x].specular = specular;
			lightVertices[curLightVerticesSize + x].metallic = metallic;
			lightVertices[curLightVerticesSize + x].roughness = roughness;
			{
				// Get some random microfacet
				float3 hDir = ImportanceSampleGGX(make_float2(hiprand_uniform(&procVertex->randState), hiprand_uniform(&procVertex->randState)), roughness, nl);

				// Calculate flesnel
				float voH = vecDot(-1 * ray.dir, hDir);
				float3 f0 = vecLerp(0.08 * make_float3(specular, specular, specular), diff, metallic);
				float3 brdf_f = Fresnel(f0, voH);

				// Reflected or Refracted
				float reflProb = lerp(length(brdf_f), 1.0f, metallic);
				float refrProb = trans;
				float3 reflDir;
				float3 refrDir;

				CURay nextRay = ray;
				float3 lightMulTerm;
				RAYTYPE nextRayType = procVertex->pathType;

				if (refrProb > 0)
				{
					bool into = vecDot(nl, norm) > 0.f;
					float nt = specular * 0.8f + 1.f;
					float nc = 1.0f;
					float nnt = into ? nc / nt : nt / nc;
					float ddn = vecDot(hDir, ray.dir);
					float cos2t = 1.f - nnt * nnt *(1.f - ddn * ddn);
					if (cos2t < 0.f)
					{
						refrProb = 0.f;
					}
					else
					{
						refrDir = normalize(ray.dir * nnt - hDir * (ddn*nnt + sqrtf(cos2t)));
					}
				}

				if (reflProb > 0)
				{
					reflDir = normalize(ray.dir - hDir * 2 * vecDot(hDir, ray.dir));
					if (vecDot(reflDir, nl) < 0.f)
						reflProb = 0.f;
				}

				// Reflected
				if (reflProb > 0 && hiprand_uniform(&procVertex->randState) < reflProb)
				{
					nextRay = CURay(ray.orig + (traceResult.dist - M_FLT_BIAS_EPSILON) * ray.dir, reflDir);
					// ShootRayResult nextRayResult = pt0_normalRay<depth + 1>(nextRay, vertices, triangles, materials, textures, randstate);

					// Microfacet specular = D*G*F / (4*NoL*NoV)
					// pdf = D * NoH / (4 * VoH)
					// (G * F * VoH) / (NoV * NoH)
					float VoH = vecDot(-1 * ray.dir, hDir);
					float NoV = vecDot(nl, -1 * ray.dir);
					float NoH = vecDot(nl, hDir);
					float NoL = vecDot(nl, reflDir);
					float G = GeometricVisibility(roughness, NoV, NoL, VoH);
					//shadeResult = vecMul((brdf_f * G * VoH) / (NoV * NoH * reflProb) , nextRayResult.light) + emissive;
					lightMulTerm = (brdf_f * G * VoH) / (NoV * NoH * reflProb);
					nextRayType = RAYTYPE_SPEC;
				}

				// Diffused or Transmited
				else
				{
					// Transmited
					if (refrProb > 0 && hiprand_uniform(&procVertex->randState) < refrProb)
					{
						nextRay = CURay(ray.orig + (traceResult.dist + M_FLT_BIAS_EPSILON) * ray.dir + refrDir * M_FLT_BIAS_EPSILON, refrDir);
						//ShootRayResult nextRayResult = pt0_normalRay<depth + 1>(nextRay, vertices, triangles, materials, textures, randstate);
						float cosine = vecDot(-1 * nl, refrDir);
						//shadeResult = (cosine * vecMul(diff, nextRayResult.light)) / (refrProb * (1 - reflProb)) + emissive;
						lightMulTerm = cosine * diff / (refrProb * (1 - reflProb));
						nextRayType = RAYTYPE_SPEC;
					}
					// Diffused
					else
					{
						float3 w = nl;
						float3 u = normalize(vecCross((fabs(w.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), w));
						float3 v = vecCross(w, u);
						u = vecCross(v, w);

						float r1 = 2.f * M_PI * hiprand_uniform(&procVertex->randState);
						float r2cos = sqrtf(hiprand_uniform(&procVertex->randState));
						float r2sin = 1.f - r2cos*r2cos;
						float3 diffDir = normalize(w * r2cos + u * r2sin * cosf(r1) + v * r2sin * sinf(r1));

						nextRay = CURay(ray.orig + traceResult.dist * ray.dir + diffDir * M_FLT_BIAS_EPSILON, diffDir);
						//ShootRayResult nextRayResult = pt0_normalRay<depth + 1>(nextRay, vertices, triangles, materials, textures, randstate);

						float VoH = vecDot(-1 * ray.dir, hDir);
						float NoV = vecDot(nl, -1 * ray.dir);
						float NoL = vecDot(nl, diffDir);
						//shadeResult = (M_PI * vecMul(Diffuse(diff, roughness, NoV, NoL, VoH), nextRayResult.light)) / ((1 - refrProb) * (1 - reflProb)) + emissive;
						lightMulTerm = M_PI * Diffuse(diff, roughness, NoV, NoL, VoH) / ((1 - refrProb) * (1 - reflProb));
						nextRayType = RAYTYPE_DIFF;
					}
				}

				procVertex->pathSample = emissive + vecMul(procVertex->pathSample, lightMulTerm);

				float pixelContrib = length(procVertex->pathOutMulTerm) * length(lightMulTerm);

				if ((procVertex->pathType == RAYTYPE_DIFF && nextRayType == RAYTYPE_SPEC) || length(emissive) > 0.f)
					pixelContrib = 0.f;

				if (hiprand_uniform(&procVertex->randState) > pixelContrib || procVertex->pathSampleDepth + 1 >= NORMALRAY_BOUND_MAX)
				{
					procVertex->isTerminated = true;
				}
				else
				{
					procVertex->pathOutMulTerm = vecMul(procVertex->pathOutMulTerm, lightMulTerm);
					procVertex->pathOutDir = nextRay.dir;
					procVertex->pathVertexPos = nextRay.orig;
					procVertex->pathType = nextRayType;
					procVertex->pathSampleDepth++;
				}
			}
		}
		else
		{
			lightVertices[curLightVerticesSize + x] = lightVertices[procVertex->pathPixel.x];
			procVertex->isTerminated = true;
		}
	}

	__global__ void pt_traceSample_kernel(RTVertex* vertices, RTTriangle* triangles, RTMaterial* materials, CURTTexture* textures, PTPathVertex** pathStream, uint activePathStreamSize)
	{
		uint x = blockIdx.x * blockDim.x + threadIdx.x;
		if (x >= activePathStreamSize || pathStream[x]->isTerminated) return;
		PTPathVertex* procVertex = pathStream[x];
		CURay ray = CURay(procVertex->pathVertexPos, procVertex->pathOutDir);
		TracePrimitiveResult traceResult;
		if (TracePrimitive(ray, traceResult, M_INF, M_FLT_BIAS_EPSILON, false))
		{
			RTTriangle* tri = &triangles[traceResult.triId];
			RTMaterial* mat = &materials[tri->matInd];
			RTVertex* v0 = &vertices[tri->vertInd0];
			RTVertex* v1 = &vertices[tri->vertInd1];
			RTVertex* v2 = &vertices[tri->vertInd2];
			float2 uv0 = make_float2(v0->tex._x, v0->tex._y);
			float2 uv1 = make_float2(v1->tex._x, v1->tex._y);
			float2 uv2 = make_float2(v2->tex._x, v2->tex._y);
			float2 uv = uv0 * traceResult.w + uv1 * traceResult.u + uv2 * traceResult.v;
			float3 n0 = V32F3(v0->norm);
			float3 n1 = V32F3(v1->norm);
			float3 n2 = V32F3(v2->norm);
			float3 norm = n0 * traceResult.w + n1 * traceResult.u + n2 * traceResult.v;

			float3 diff;
			float3 emissive;
			float trans;
			float specular;
			float metallic;
			float roughness;
			float anisotropic;
			float sheen;
			float sheenTint;
			float clearcoat;
			float clearcoatGloss;
			GetMaterialColors(mat, uv, textures, diff, norm, emissive, trans, specular, metallic, roughness
				, anisotropic, sheen, sheenTint, clearcoat, clearcoatGloss);
			float3 nl = vecDot(norm, ray.dir) < 0.f ? norm : -1 * norm;
			{
				// Get some random microfacet
				float3 hDir = ImportanceSampleGGX(make_float2(hiprand_uniform(&procVertex->randState), hiprand_uniform(&procVertex->randState)), roughness, nl);

				// Calculate flesnel
				float voH = vecDot(-1 * ray.dir, hDir);
				float3 f0 = vecLerp(0.08 * make_float3(specular, specular, specular), diff, metallic);
				float3 brdf_f = Fresnel(f0, voH);

				// Reflected or Refracted
				float reflProb = lerp(length(brdf_f), 1.0f, metallic);
				float refrProb = trans;
				float3 reflDir;
				float3 refrDir;

				CURay nextRay = ray;
				float3 lightMulTerm;
				RAYTYPE nextRayType = procVertex->pathType;

				if (refrProb > 0)
				{
					bool into = vecDot(nl, norm) > 0.f;
					float nt = specular * 0.8f + 1.f;
					float nc = 1.0f;
					float nnt = into ? nc / nt : nt / nc;
					float ddn = vecDot(hDir, ray.dir);
					float cos2t = 1.f - nnt * nnt *(1.f - ddn * ddn);
					if (cos2t < 0.f)
					{
						refrProb = 0.f;
					}
					else
					{
						refrDir = normalize(ray.dir * nnt - hDir * (ddn*nnt + sqrtf(cos2t)));
					}
				}

				if (reflProb > 0)
				{
					reflDir = normalize(ray.dir - hDir * 2 * vecDot(hDir, ray.dir));
					if (vecDot(reflDir, nl) < 0.f)
						reflProb = 0.f;
				}

				// Reflected
				if (reflProb > 0 && hiprand_uniform(&procVertex->randState) < reflProb)
				{
					nextRay = CURay(ray.orig + (traceResult.dist - M_FLT_BIAS_EPSILON) * ray.dir, reflDir);
					// ShootRayResult nextRayResult = pt0_normalRay<depth + 1>(nextRay, vertices, triangles, materials, textures, randstate);

					// Microfacet specular = D*G*F / (4*NoL*NoV)
					// pdf = D * NoH / (4 * VoH)
					// (G * F * VoH) / (NoV * NoH)
					float VoH = vecDot(-1 * ray.dir, hDir);
					float NoV = vecDot(nl, -1 * ray.dir);
					float NoH = vecDot(nl, hDir);
					float NoL = vecDot(nl, reflDir);
					float G = GeometricVisibility(roughness, NoV, NoL, VoH);
					//shadeResult = vecMul((brdf_f * G * VoH) / (NoV * NoH * reflProb) , nextRayResult.light) + emissive;
					lightMulTerm = (brdf_f * G * VoH) / (NoV * NoH * reflProb);
					nextRayType = RAYTYPE_SPEC;
				}

				// Diffused or Transmited
				else
				{
					// Transmited
					if (refrProb > 0 && hiprand_uniform(&procVertex->randState) < refrProb)
					{
						nextRay = CURay(ray.orig + (traceResult.dist + M_FLT_BIAS_EPSILON) * ray.dir + refrDir * M_FLT_BIAS_EPSILON, refrDir);
						//ShootRayResult nextRayResult = pt0_normalRay<depth + 1>(nextRay, vertices, triangles, materials, textures, randstate);
						float cosine = vecDot(-1 * nl, refrDir);
						//shadeResult = (cosine * vecMul(diff, nextRayResult.light)) / (refrProb * (1 - reflProb)) + emissive;
						lightMulTerm = cosine * diff / (refrProb * (1 - reflProb));
						nextRayType = RAYTYPE_SPEC;
					}
					// Diffused
					else
					{
						float3 w = nl;
						float3 u = normalize(vecCross((fabs(w.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), w));
						float3 v = vecCross(w, u);
						u = vecCross(v, w);

						float r1 = 2.f * M_PI * hiprand_uniform(&procVertex->randState);
						float r2cos = sqrtf(hiprand_uniform(&procVertex->randState));
						float r2sin = 1.f - r2cos*r2cos;
						float3 diffDir = normalize(w * r2cos + u * r2sin * cosf(r1) + v * r2sin * sinf(r1));

						nextRay = CURay(ray.orig + traceResult.dist * ray.dir + diffDir * M_FLT_BIAS_EPSILON, diffDir);
						//ShootRayResult nextRayResult = pt0_normalRay<depth + 1>(nextRay, vertices, triangles, materials, textures, randstate);

						float VoH = vecDot(-1 * ray.dir, hDir);
						float NoV = vecDot(nl, -1 * ray.dir);
						float NoL = vecDot(nl, diffDir);
						//shadeResult = (M_PI * vecMul(Diffuse(diff, roughness, NoV, NoL, VoH), nextRayResult.light)) / ((1 - refrProb) * (1 - reflProb)) + emissive;
						lightMulTerm = M_PI * Diffuse(diff, roughness, NoV, NoL, VoH) / ((1 - refrProb) * (1 - reflProb));
						nextRayType = RAYTYPE_DIFF;
					}
				}

				procVertex->pathSample = procVertex->pathSample + vecMul(emissive , procVertex->pathOutMulTerm);

				procVertex->origDiff = diff;
				procVertex->pathInDir = -1 * ray.dir;
				procVertex->origNorm = nl;
				procVertex->origRoughness = roughness;
				procVertex->origMetallic = metallic;
				procVertex->origSpecular = specular;
				procVertex->origTrans = trans;
				procVertex->pathInMulTerm = procVertex->pathOutMulTerm;

				float pixelContrib = length(procVertex->pathOutMulTerm) * length(lightMulTerm);

				if ((procVertex->pathType == RAYTYPE_DIFF && nextRayType == RAYTYPE_SPEC) || length(emissive) > 0.f)
					pixelContrib = 0.f;

				if (hiprand_uniform(&procVertex->randState) > pixelContrib || procVertex->pathSampleDepth + 1 >= NORMALRAY_BOUND_MAX)
				{
					procVertex->pathAccumSample = procVertex->pathAccumSample + procVertex->pathSample;
					procVertex->pathSampleN++;
					procVertex->isTerminated = true;
				}
				else
				{
					procVertex->pathOutMulTerm = vecMul(procVertex->pathOutMulTerm, lightMulTerm);
					procVertex->pathOutDir = nextRay.dir;
					procVertex->pathSampleDepth++;
				}
				procVertex->pathVertexPos = nextRay.orig;
				procVertex->pathType = nextRayType;

			}
		}
		else
		{
			procVertex->pathAccumSample = procVertex->pathAccumSample + procVertex->pathSample;
			procVertex->pathSampleN++;
			procVertex->isTerminated = true;
		}
	}

	__global__ void pt_genLightPathQueue_kernel(uint32 frameN, uint32 hashedFrameN, uint* lightTri, uint lightTriN, RTVertex* vertices,
		RTTriangle* triangles, RTMaterial* materials, CURTTexture* textures, PTPathVertex* pathQueue, uint pathQueueCap, LightVertex* lightVertices, uint curLightVerticesSize)
	{
		uint x = blockIdx.x * blockDim.x + threadIdx.x;
		if (x > pathQueueCap) return;

		hiprandState randstate;
		hiprand_init(hashedFrameN + x, 0, 0, &randstate);

		uint lightSourceId = hiprand_uniform(&randstate) * lightTriN;
		float lightW = hiprand_uniform(&randstate);
		float lightU = hiprand_uniform(&randstate);
		if (lightW + lightU > 1.0f)
		{
			lightW = 1.f - lightW;
			lightU = 1.f - lightU;
		}
		float lightV = 1.f - lightW - lightU;

		uint triId = lightTri[lightSourceId];
		RTTriangle* tri = &triangles[triId];
		RTMaterial* mat = &materials[tri->matInd];
		RTVertex* v0 = &vertices[tri->vertInd0];
		RTVertex* v1 = &vertices[tri->vertInd1];
		RTVertex* v2 = &vertices[tri->vertInd2];
		float2 uv0 = make_float2(v0->tex._x, v0->tex._y);
		float2 uv1 = make_float2(v1->tex._x, v1->tex._y);
		float2 uv2 = make_float2(v2->tex._x, v2->tex._y);
		float2 uv = uv0 * lightW + uv1 * lightU + uv2 * lightV;
		float3 n0 = V32F3(v0->norm);
		float3 n1 = V32F3(v1->norm);
		float3 n2 = V32F3(v2->norm);
		float3 triNorm = n0 * lightW + n1 * lightU + n2 * lightV;
		float3 triPos = V32F3(v0->pos) * lightW + V32F3(v1->pos) * lightU + V32F3(v2->pos) * lightV;

		float3 diff;
		float3 emissive;
		float trans;
		float specular;
		float metallic;
		float roughness;
		float anisotropic;
		float sheen;
		float sheenTint;
		float clearcoat;
		float clearcoatGloss;
		GetMaterialColors(mat, uv, textures, diff, triNorm, emissive, trans, specular, metallic, roughness
			, anisotropic, sheen, sheenTint, clearcoat, clearcoatGloss);

		float3 w = triNorm;
		float3 u = normalize(vecCross((fabs(w.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), w));
		float3 v = vecCross(w, u);
		u = vecCross(v, w);

		float r1 = 2.f * M_PI * hiprand_uniform(&randstate);
		float r2cos = sqrtf(hiprand_uniform(&randstate));
		float r2sin = 1.f - r2cos*r2cos;
		float3 diffDir = normalize(w * r2cos + u * r2sin * cosf(r1) + v * r2sin * sinf(r1));

		pathQueue[x] = PTPathVertex(false, make_uint2(curLightVerticesSize + x, 0), diffDir
			, triPos + M_FLT_BIAS_EPSILON * triNorm, RAYTYPE_LIGHT, randstate);
		pathQueue[x].pathSample = emissive;

		lightVertices[curLightVerticesSize + x].irrad = emissive;
		lightVertices[curLightVerticesSize + x].irradDir = make_float3(0.f, 0.f, 0.f);
		lightVertices[curLightVerticesSize + x].norm = triNorm;
		lightVertices[curLightVerticesSize + x].pos = triPos;
		lightVertices[curLightVerticesSize + x].diff = diff;
		lightVertices[curLightVerticesSize + x].emissive = emissive;
		lightVertices[curLightVerticesSize + x].specular = specular;
		lightVertices[curLightVerticesSize + x].metallic = metallic;
		lightVertices[curLightVerticesSize + x].roughness = roughness;
	}

	__global__ void pt_genPathQueue_kernel(float3 camPos, float3 camDir, float3 camUp, float3 camRight, float fov,
		float width, float height, uint32 frameN, uint32 hashedFrameN, PTPathVertex* pathQueue)
	{
		uint x = blockIdx.x * blockDim.x + threadIdx.x;
		uint y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x >= width || y >= height) return;

		uint ind = (y * width + x);

		float u = (2.f * ((float)x + 0.5f) / width - 1.f) * tan(fov * 0.5f) * width / height;
		float v = (2.f * ((float)y + 0.5f) / height - 1.f) * tan(fov * 0.5f);

		hiprandState randstate;
		hiprand_init(hashedFrameN + ind, 0, 0, &randstate);
		float au = u + (hiprand_uniform(&randstate) - 0.5f) / height * tan(fov * 0.5f);
		float av = v + (hiprand_uniform(&randstate) - 0.5f) / height * tan(fov * 0.5f);

		float3 dir = normalize(camRight * au + camUp * av + camDir);

		pathQueue[ind] = PTPathVertex(false, make_uint2(x,y), dir, camPos, RAYTYPE_EYE, randstate);
	}

	__device__ float3 GetShadingResult(const float3& lightOutDir, const float3& lightInDir, const float3& lightInIrrad, const float3& norm,
		const float3& diff, const float metallic, const float roughness, const float specular, const float2 diffspec)
	{
		if (vecDot(norm, lightInDir) <= 0.f)
			return make_float3(0.f, 0.f, 0.f);

		float3 h = normalize(lightOutDir + lightInDir);

		float voH = vecDot(lightOutDir, h);
		float noV = vecDot(norm, lightOutDir);
		float noH = vecDot(norm, h);
		float noL = vecDot(norm, lightInDir);
		float3 f0 = vecLerp(0.08f * specular * make_float3(1.f, 1.f, 1.f), diff, metallic);
		float3 brdf_f = Fresnel(f0, voH);
		//float g = GeometricVisibility(roughness, noV, noL, voH);
		float d = D_GGX(roughness, noH);
		float v = Vis_SmithJointApprox(roughness, noV, noL);
		// Microfacet specular = D*G*F / (4*NoL*NoV)
		float3 specIrrad = d*v*brdf_f;// vecMul(d*g*brdf_f / (4.f * noV), lightInIrrad);
		float3 diffIrrad = vecMul((make_float3(1.f, 1.f, 1.f) - brdf_f), Diffuse(diff, roughness, noV, noL, voH));//vecMul((make_float3(1.f, 1.f, 1.f) - brdf_f), diff / M_PI);
		return vecMul(lightInIrrad*noL, diffspec.y*specIrrad + diffspec.x*diffIrrad);
	}

	__device__ void  GetLightFromRandLightVertices(float3 pos, float3 norm, LightVertex* lightVertices, uint lightVerticesSize, hiprandState* randstate, float3& irrad, float3& irradDir)
	{
		//LightVertex dummy;
		//dummy.diff = make_float3(1.f, 1.f, 1.f);
		//dummy.irrad = make_float3(1.f, 0.f, 0.f);
		//dummy.pos = make_float3(0.f, 0.f, 0.f);
		//dummy.norm = dummy.irradDir = normalize(pos - dummy.pos);
		//dummy.roughness = 0.5f;
		//dummy.specular = 0.5f;
		//dummy.metallic = 0.f;

		irrad = make_float3(0.f, 0.f, 0.f);
		uint lightVert = hiprand_uniform(randstate) * lightVerticesSize;
		LightVertex* lightVertex = &lightVertices[lightVert];
		float3 toLightVertexDir = normalize(lightVertex->pos - pos);
		float toLightVertexDist = length(lightVertex->pos - pos);

		CURay toLightVertex(pos, toLightVertexDir);
		TracePrimitiveResult traceResult;
		if (length(lightVertex->irrad) > 0.f && vecDot(norm, toLightVertexDir) > 0.f &&
			!TracePrimitive(toLightVertex, traceResult, toLightVertexDist - M_FLT_BIAS_EPSILON, M_FLT_BIAS_EPSILON, false))
		{
			if (length(lightVertex->irradDir) > M_FLT_EPSILON)
				irrad = GetShadingResult(-1 * toLightVertexDir, lightVertex->irradDir, lightVertex->irrad, lightVertex->norm
				, lightVertex->diff, lightVertex->metallic, lightVertex->roughness, lightVertex->specular, make_float2(1.f, 1.f)) + lightVertex->emissive;
			else
				irrad = lightVertex->irrad;
			irrad = irrad;
			irradDir = toLightVertexDir;
		}
	}


	__global__ void pt_connectEyeLightPath_kernel(PTPathVertex** eyeStream, uint eyeStreamSize, LightVertex* lightVertices, uint lightVerticesSize)
	{
		uint ind = blockIdx.x * blockDim.x + threadIdx.x;
		if (ind >= eyeStreamSize) return;

		PTPathVertex* eyePath = eyeStream[ind];
		float3 lightFromLightVertex = make_float3(0.f, 0.f, 0.f);
		float3 toLightVertexDir = make_float3(0.f, 0.f, 0.f);
		GetLightFromRandLightVertices(eyePath->pathVertexPos + eyePath->origNorm * M_FLT_BIAS_EPSILON, eyePath->origNorm
			, lightVertices, lightVerticesSize, &eyePath->randState, lightFromLightVertex, toLightVertexDir);
		float3 lightContribFromLightVertex = vecMax(make_float3(0.f, 0.f, 0.f)
			, GetShadingResult(eyePath->pathInDir, toLightVertexDir, lightFromLightVertex, eyePath->origNorm
			, eyePath->origDiff, eyePath->origMetallic, eyePath->origRoughness, eyePath->origSpecular
			, make_float2(1.f - eyePath->origTrans, 1.f)));

		if (length(lightContribFromLightVertex) > 0.f)
		{
			eyePath->pathAccumSample = eyePath->pathAccumSample + vecMul(lightContribFromLightVertex, eyePath->pathInMulTerm);
			eyePath->pathSampleN += 4;
		}
	}

	__global__ void pt_assignPathStream_kernel(PTPathVertex** pathStream, uint pathStreamSize, PTPathVertex* pathQueue, uint pathQueueCur, uint pathQueueSize, uint assignableSlot)
	{
		uint ind = blockIdx.x * blockDim.x + threadIdx.x;
		if (ind < assignableSlot)
		{
			int pathStreamInd = pathStreamSize + ind;
			int pathQueueInd = pathQueueCur + ind;
			PTPathVertex* assignSample = nullptr;
			if (pathQueueInd < pathQueueSize)
			{
				assignSample = &pathQueue[pathQueueInd];
			}
			pathStream[pathStreamInd] = assignSample;
		}
	}

	__global__ void pt_applyPathQueueResult_kernel(PTPathVertex* pathQueue, uint pathQueueSize, uint width, uint height, uint frameN, float* result, float* accResult)
	{
		uint x = blockIdx.x * blockDim.x + threadIdx.x;

		if (x >= pathQueueSize) return;

		// add calculating sample to the result
		if (!pathQueue[x].isTerminated)
		{
			pathQueue[x].pathAccumSample = pathQueue[x].pathAccumSample + pathQueue[x].pathSample;
			pathQueue[x].pathSampleN++;
		}

		if (pathQueue[x].pathSampleN > 0)
		{
			float3 sampleResult = pathQueue[x].pathAccumSample / (float)pathQueue[x].pathSampleN;
			float resultInf = 1.f / (float)(frameN + 1);
			float oldInf = 1.f - resultInf;
			uint ind = pathQueue[x].pathPixel.y * width + pathQueue[x].pathPixel.x;
			result[ind * 3] = max(resultInf * sampleResult.x + oldInf * result[ind * 3], 0.f);
			result[ind * 3 + 1] = max(resultInf * sampleResult.y + oldInf * result[ind * 3 + 1], 0.f);
			result[ind * 3 + 2] = max(resultInf * sampleResult.z + oldInf * result[ind * 3 + 2], 0.f);
		}
	}

	void CleanMem()
	{
		freeLightPathMem();
		freeStreamMem();
		freeAllBVHCudaMem();
		CUFREE(g_devResultData);
		CUFREE(g_devAccResultData);
	}

	//struct ray_greater_compare
	//{
	//	__hd__ bool operator()(const PTPathVertex* vert1, const PTPathVertex* vert2)
	//	{
	//		int vert1Score = (vert1->pathOutDir.x > 0) + (vert1->pathOutDir.y > 0) + (vert1->pathOutDir.z > 0);
	//		int vert2Score = (vert2->pathOutDir.x > 0) + (vert2->pathOutDir.y > 0) + (vert2->pathOutDir.z > 0);
	//		return vert1Score > vert2Score;
	//	}
	//};

	struct is_terminated
	{
		__hd__ bool operator()(const PTPathVertex* vert)
		{
			return vert->isTerminated;
		}
	};

	struct is_connectToLightPath
	{
		__hd__ bool operator()(const PTPathVertex* vert)
		{
			return vert->pathType == RAYTYPE_DIFF;
		}
	};

	bool Render(NPMathHelper::Vec3 camPos, NPMathHelper::Vec3 camDir, NPMathHelper::Vec3 camUp, float fov, RTScene* scene
		, float width, float height, float* result)
	{
		// Check and allocate everything
		if (!scene || !scene->GetCompactBVH()->IsValid())
			return false;

		NPMathHelper::Vec3 camRight = camDir.cross(camUp).normalize();
		camUp = camRight.cross(camDir).normalize();

		g_matLastCamMat = g_matCurCamMat;
		g_matCurCamMat = NPMathHelper::Mat4x4::lookAt(camPos, camPos + camDir, camUp);
		g_uCurFrameN = (g_matLastCamMat != g_matCurCamMat) ? 0 : g_uCurFrameN + 1;

		if (!g_bIsCudaInit || scene->GetIsCudaDirty())
		{
			g_matLastCamMat = g_matCurCamMat;
			g_uCurFrameN = 0;
			initAllSceneCudaMem(scene);
			allocateStreamMem(width * height);
			allocateLightPathMem();
			updateLightTriCudaMem(scene);

			size_t mem_tot;
			size_t mem_free;
			hipMemGetInfo(&mem_free, &mem_tot);
			std::cout << "Memory Used : " << mem_tot-mem_free << "/" << mem_tot << " -> Free " << mem_free << std::endl;
		}
		else if (scene->GetIsCudaMaterialDirty())
		{
			updateAllSceneMaterialsCudaMem(scene);
			updateLightTriCudaMem(scene);
			g_uCurFrameN = 0;
		}

		if (!g_bIsCudaInit)
			return false;

		if (!g_devResultData || !g_devAccResultData || g_resultDataSize != (sizeof(float) * 3 * width * height))
		{
			g_resultDataSize = sizeof(float) * 3 * width * height;
			CUFREE(g_devResultData);
			hipMalloc((void**)&g_devResultData, g_resultDataSize);
			CUFREE(g_devAccResultData);
			hipMalloc((void**)&g_devAccResultData, g_resultDataSize);
		}

		float3 f3CamPos = V32F3(camPos);
		float3 f3CamUp = V32F3(camUp);
		float3 f3CamDir = V32F3(camDir);
		float3 f3CamRight = V32F3(camRight);
		dim3 block1(BLOCK_SIZE*BLOCK_SIZE, 1, 1);
		dim3 block2(BLOCK_SIZE, BLOCK_SIZE, 1);
		dim3 renderGrid(ceil(width / (float)block2.x), ceil(height / (float)block2.y), 1);

		// light paths
		if (g_uCurFrameN % 3 == 0)
		{
			uint lightPathStreamSizeCap = min((uint)PATHSTREAM_SIZE, (uint)(LIGHTVERTEX_N / LIGHTRAY_BOUND_MAX));
			pt_genLightPathQueue_kernel << < dim3(ceil((float)lightPathStreamSizeCap / (float)block1.x), 1, 1), block1 >> >
				(g_uCurFrameN, WangHash(g_uCurFrameN), g_devLightTri, g_lightTriN, g_devVertices, g_devTriangles, g_devMaterials, g_devTextures, g_devPathQueue, lightPathStreamSizeCap
				, g_devLightVertices, 0);

			uint activePathStreamSize = 0;
			g_uLightVerticesSize = lightPathStreamSizeCap;
			g_uPathQueueCur = 0;
			while (g_uPathQueueCur < lightPathStreamSizeCap || activePathStreamSize > 0)
			{
				uint tempActivePathStreamSize = activePathStreamSize;
				int assignableStreamSlot = min(lightPathStreamSizeCap - activePathStreamSize, lightPathStreamSizeCap - g_uPathQueueCur);
				if (assignableStreamSlot > 0)
				{
					pt_assignPathStream_kernel << < dim3(ceil((float)assignableStreamSlot / (float)block1.x), 1, 1), block1 >> >(g_devPathStream, activePathStreamSize, g_devPathQueue, g_uPathQueueCur
						, g_uLightVerticesSize, assignableStreamSlot);
				}
				//readjust activePathStreamSize
				activePathStreamSize += assignableStreamSlot;
				g_uPathQueueCur += assignableStreamSlot;

				pt_traceLight_kernel << < dim3(ceil((float)activePathStreamSize / (float)block1.x), 1, 1), block1 >> > (g_devVertices, g_devTriangles, g_devMaterials, g_devTextures, g_devPathStream, activePathStreamSize
					, g_devLightVertices, g_uLightVerticesSize);
				g_uLightVerticesSize += activePathStreamSize;
				//compact pathstream and find activePathStreamSize value
				PTPathVertex** compactedStreamEndItr = thrust::remove_if(thrust::device, g_devPathStream, g_devPathStream + activePathStreamSize, is_terminated());
				activePathStreamSize = compactedStreamEndItr - g_devPathStream;
			}
			std::cout << "Generated light vertices size: " << g_uLightVerticesSize << std::endl;
		}

		// eye paths
		pt_genPathQueue_kernel << < renderGrid, block2 >> > (f3CamPos, f3CamDir, f3CamUp, f3CamRight, fov, width, height
			, g_uCurFrameN, WangHash(g_uCurFrameN), g_devPathQueue);

		uint activePathStreamSize = 0;
		g_uPathQueueCur = 0;
		while (g_uPathQueueCur < g_uPathQueueSize || activePathStreamSize > 0)
		{
			uint tempActivePathStreamSize = activePathStreamSize;
			int assignableStreamSlot = min((uint)PATHSTREAM_SIZE - activePathStreamSize, g_uPathQueueSize - g_uPathQueueCur);
			if (assignableStreamSlot > 0)
				pt_assignPathStream_kernel << < dim3(ceil((float)assignableStreamSlot / (float)block1.x), 1, 1), block1 >> >(g_devPathStream, activePathStreamSize, g_devPathQueue, g_uPathQueueCur
				, g_uPathQueueSize, assignableStreamSlot);

			//readjust activePathStreamSize
			activePathStreamSize += assignableStreamSlot;
			g_uPathQueueCur += assignableStreamSlot;

			//tracing process
			pt_traceSample_kernel << < dim3(ceil((float)activePathStreamSize / (float)block1.x), 1, 1), block1 >> > (g_devVertices, g_devTriangles, g_devMaterials, g_devTextures, g_devPathStream, activePathStreamSize);

			//compact pathstream and find activePathStreamSize value
			PTPathVertex** compactedStreamEndItr = thrust::remove_if(thrust::device, g_devPathStream, g_devPathStream + activePathStreamSize, is_terminated());
			activePathStreamSize = compactedStreamEndItr - g_devPathStream;

			//gen connectionpathstream
			PTPathVertex** conPathStreamEndItr = thrust::copy_if(thrust::device, g_devPathStream, g_devPathStream + activePathStreamSize, g_devEyeLightConPathStream, is_connectToLightPath());
			uint activeConPathStreamSize = conPathStreamEndItr - g_devEyeLightConPathStream;

			//connect eye and light path stream
			if (activeConPathStreamSize > 0)
			{
				pt_connectEyeLightPath_kernel << < dim3(ceil((float)activeConPathStreamSize / (float)block1.x), 1, 1), block1 >> >
					(g_devEyeLightConPathStream, activeConPathStreamSize, g_devLightVertices, g_uLightVerticesSize);
			}

		}
		pt_applyPathQueueResult_kernel << < dim3(ceil((float)g_uPathQueueSize / (float)block1.x), 1, 1), block1 >> >(g_devPathQueue, g_uPathQueueSize, width, height, g_uCurFrameN, g_devResultData, g_devAccResultData);

		// Copy result to host
		hipMemcpy(result, g_devResultData, g_resultDataSize, hipMemcpyDeviceToHost);
		return true;
	}
}