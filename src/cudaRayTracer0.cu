#include "hip/hip_runtime.h"
#include "cudahelper.h"
#include "raytracer.h"
#include "mathhelper.h"

#include <tbb/parallel_for.h>
#include <tbb/blocked_range.h>

#define BLOCK_SIZE 32

texture<float4, 1, hipReadModeElementType> g_bvhMinMaxBounds;
texture<uint1, 1, hipReadModeElementType> g_bvhOffsetTriStartN;
texture<float4, 1, hipReadModeElementType> g_triIntersectionData;
RTTriangle* g_devTriangles = nullptr;
RTMaterial* g_devMaterials = nullptr;
float* g_devResultData = nullptr;
float4* g_devBVHMinMaxBounds = nullptr;
uint1* g_devBVHOffsetTriStartN = nullptr;
float4* g_devTriIntersectionData = nullptr;

bool g_bIsCudaInit = false;
size_t g_resultDataSize = 0;

struct CURay
{
	float3 orig;
	float3 dir;
	__hd__ CURay(float3 _orig, float3 _dir) : orig(_orig), dir(_dir) {}
	__hd__ float IntersectAABB(const float3& _min, const float3& _max)
	{
		float3 modDir = dir;
		dir.x = escapeZero(dir.x, M_EPSILON);
		dir.y = escapeZero(dir.y, M_EPSILON);
		dir.z = escapeZero(dir.z, M_EPSILON);
		float3 tmin = (_min - orig) / modDir;
		float3 tmax = (_max - orig) / modDir;
		float3 real_min = vecMin(tmin, tmax);
		float3 real_max = vecMax(tmin, tmax);
		float minmax = min(min(real_max.x, real_max.y), real_max.z);
		float maxmin = max(max(real_min.x, real_min.y), real_min.z);
		if (minmax >= maxmin)
			return (maxmin > M_EPSILON) ? maxmin : 0;
		return M_INF;
	}

	template<bool cullback = true>
	__hd__ float IntersectTri(const float3& _p0, const float3& _e0, const float3& _e1, float& w, float& u, float& v)
	{
		if (cullback && vecDot(vecCross(_e0, _e1), dir) > 0.f)
			return M_INF;
		float3 de2 = vecCross(dir, _e1);
		float divisor = vecDot(de2, _e0);
		if (fabs(divisor) < M_EPSILON)
			return M_INF;
		float3 t = orig - _p0;
		float3 te1 = vecCross(t, _e0);
		float rT = vecDot(te1, _e1) / divisor;
		if (rT < 0.f)
			return M_INF;
		u = vecDot(de2, t) / divisor;
		if (u < 0.f || u > 1.f)
			return M_INF;
		v = vecDot(te1, dir) / divisor;
		if (v < 0.f || (u + v) > 1.f)
			return M_INF;
		w = 1 - u - v;
		return rT;
	}
};

__global__ void pt0_kernel(float3 camPos, float3 camDir, float3 camUp, float3 camRight, float fov,
	float width, float height, uint bvhNodeN, float* result)
{
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;
	uint ind = (y * width + x) * 3;
	int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	float u = (2.f * ((float)x + 0.5f) / width - 1.f) * tan(fov * 0.5f) * width / height;
	float v = (2.f * ((float)y + 0.5f) / height - 1.f) * tan(fov * 0.5f);
	float3 dir = normalize(camRight * u + camUp * v + camDir);
	CURay ray(camPos, dir);

	{
		float minIntersect = M_INF;
		uint32 tracedTriId = 0;
		float w, u, v;
		uint32 traceCmd[128];
		traceCmd[0] = 0;
		int32 traceCmdPointer = 0;
		while (traceCmdPointer >= 0)
		{
			uint32 curInd = traceCmd[traceCmdPointer--];
			float4 boundMin = tex1Dfetch(g_bvhMinMaxBounds, curInd * 2);
			float4 boundMax = tex1Dfetch(g_bvhMinMaxBounds, curInd * 2 + 1);
			float min = ray.IntersectAABB(make_float3(boundMin.x, boundMin.y, boundMin.z),
				make_float3(boundMax.x, boundMax.y, boundMax.z));
			if (min >= 0 && min < minIntersect)
			{
				uint1 offOrTs = tex1Dfetch(g_bvhOffsetTriStartN, curInd * 2);
				uint1 tN = tex1Dfetch(g_bvhOffsetTriStartN, curInd * 2 + 1);
				if (tN.x == 0)
				{
					if (traceCmdPointer < 126)
					{
						traceCmd[++traceCmdPointer] = curInd + 1;
						traceCmd[++traceCmdPointer] = curInd + offOrTs.x;
					}
				}
				else
				{
					for (uint32 i = offOrTs.x; i < offOrTs.x + tN.x; i++)
					{
						float _w, _u, _v;
						float4 p0 = tex1Dfetch(g_triIntersectionData, i * 3);
						float4 e0 = tex1Dfetch(g_triIntersectionData, i * 3 + 1);
						float4 e1 = tex1Dfetch(g_triIntersectionData, i * 3 + 2);
						float triIntersect = ray.IntersectTri(make_float3(p0.x, p0.y, p0.z),
							make_float3(e0.x, e0.y, e0.z), make_float3(e1.x, e1.y, e1.z),
							_w, _u, _v);
						if (triIntersect >= 0 && triIntersect < minIntersect)
						{
							minIntersect = triIntersect;
							tracedTriId = i;
							w = _w; u = _u; v = _v;
						}
					}
				}
			}
		}

		if (minIntersect < M_INF)
		{
			result[ind] = w;
			result[ind + 1] = u;
			result[ind + 2] = v;
		}
		else
		{
			result[ind] = dir.x;
			result[ind + 1] = dir.y;
			result[ind + 2] = dir.z;
		}
	}
}

float4 V32F4(const NPMathHelper::Vec3& vec3)
{
	return make_float4(vec3._x, vec3._y, vec3._z, 0.f);
}

template<class T, int dim, enum hipTextureReadMode readMode>
void BindCudaTexture(texture<T, dim, readMode> *tex, void* data, size_t size)
{
	tex->normalized = false;
	tex->filterMode = hipFilterModePoint;
	tex->addressMode[0] = hipAddressModeWrap;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<T>();
	HANDLE_ERROR(hipBindTexture(0, *tex, data, channelDesc, size));
}

bool cudaPT0Render(float3 camPos, float3 camDir, float3 camUp, float fov, RTScene* scene
	, float width, float height, float* result)
{
	// Check and allocate everything
	if (!scene || !scene->GetCompactBVH()->IsValid())
		return false;
	if (!g_bIsCudaInit || scene->GetIsCudaDirty())
	{
		if (g_bIsCudaInit)
		{
			HANDLE_ERROR(hipUnbindTexture(g_bvhMinMaxBounds));
			HANDLE_ERROR(hipUnbindTexture(g_bvhOffsetTriStartN));
			HANDLE_ERROR(hipUnbindTexture(g_triIntersectionData));
		}

		uint triSize = scene->m_pTriangles.size();
		RTTriangle* tempTriangles = new RTTriangle[triSize];
		float4* tempTriIntersectionData = new float4[triSize * 3];
		{
			auto f = [&](const tbb::blocked_range< int >& range) {
				for (unsigned int i = range.begin(); i < range.end(); i++)
				{
					tempTriIntersectionData[i * 3] = V32F4((*scene->GetTriIntersectData())[i * 3]);
					tempTriIntersectionData[i * 3 + 1] = V32F4(((*scene->GetTriIntersectData())[i * 3 + 1] - (*scene->GetTriIntersectData())[i * 3]));
					tempTriIntersectionData[i * 3 + 2] = V32F4(((*scene->GetTriIntersectData())[i * 3 + 2] - (*scene->GetTriIntersectData())[i * 3]));
					tempTriangles[i] = scene->m_pTriangles[i];
				}
			};
			tbb::parallel_for(tbb::blocked_range< int >(0, triSize), f);
		}

		uint bvhNodeN = scene->GetCompactBVH()->nodeN;
		float4* tempBVHMinMaxBounds = new float4[bvhNodeN * 2];
		uint1* tempBVHOffsetTriStartN = new uint1[bvhNodeN * 2];
		{
			auto f = [&](const tbb::blocked_range< int >& range) {
				for (unsigned int i = range.begin(); i < range.end(); i++)
				{
					tempBVHMinMaxBounds[i * 2] = V32F4(scene->GetCompactBVH()->bounds[i].minPoint);
					tempBVHMinMaxBounds[i * 2 + 1] = V32F4(scene->GetCompactBVH()->bounds[i].maxPoint);
					tempBVHOffsetTriStartN[i * 2].x = scene->GetCompactBVH()->offOrTSTN[i * 2];
					tempBVHOffsetTriStartN[i * 2 + 1].x = scene->GetCompactBVH()->offOrTSTN[i * 2 + 1];
				}
			};
			tbb::parallel_for(tbb::blocked_range< int >(0, bvhNodeN), f);
		}
		CUFREE(g_devBVHMinMaxBounds);
		CUFREE(g_devBVHOffsetTriStartN);
		CUFREE(g_devTriIntersectionData);
		CUFREE(g_devTriangles);
		CUFREE(g_devMaterials);

		// Create Dev Data
		hipMalloc((void**)&g_devTriangles, sizeof(RTTriangle) * triSize);
		hipMalloc((void**)&g_devTriIntersectionData, sizeof(float4) * triSize * 3);
		hipMalloc((void**)&g_devBVHOffsetTriStartN, sizeof(uint1) * bvhNodeN * 2);
		hipMalloc((void**)&g_devBVHMinMaxBounds, sizeof(float4) * bvhNodeN * 2);

		// MemCpy Dev Data
		hipMemcpy(g_devTriangles, tempTriangles, sizeof(RTTriangle) * triSize, hipMemcpyHostToDevice);
		hipMemcpy(g_devTriIntersectionData, tempTriIntersectionData, sizeof(float4) * triSize * 3, hipMemcpyHostToDevice);
		hipMemcpy(g_devBVHOffsetTriStartN, tempBVHOffsetTriStartN, sizeof(uint1) * bvhNodeN * 2, hipMemcpyHostToDevice);
		hipMemcpy(g_devBVHMinMaxBounds, tempBVHMinMaxBounds, sizeof(float4) * bvhNodeN * 2, hipMemcpyHostToDevice);

		// Del Temp Data
		DEL_ARRAY(tempBVHOffsetTriStartN);
		DEL_ARRAY(tempBVHMinMaxBounds);
		DEL_ARRAY(tempTriIntersectionData);
		DEL_ARRAY(tempTriangles);

		// Bind Dev Data To Texture
		BindCudaTexture(&g_bvhMinMaxBounds, g_devBVHMinMaxBounds, sizeof(float4) * bvhNodeN * 2);
		BindCudaTexture(&g_bvhOffsetTriStartN, g_devBVHOffsetTriStartN, sizeof(uint1) * bvhNodeN * 2);
		BindCudaTexture(&g_triIntersectionData, g_devTriIntersectionData, sizeof(float4) * triSize * 3);



		g_bIsCudaInit = true;
		scene->SetIsCudaDirty();
	}

	if (!g_bIsCudaInit)
		return false;

	if (!g_devResultData || g_resultDataSize != (sizeof(float) * 3 * width * height))
	{
		g_resultDataSize = sizeof(float) * 3 * width * height;
		CUFREE(g_devResultData);
		hipMalloc((void**)&g_devResultData, g_resultDataSize);
	}

	float3 camRight = normalize(vecCross(camDir, camUp));
	camUp = normalize(vecCross(camRight, camDir));

	// Kernel go here
	dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 grid(width / block.x, height / block.y, 1);
	pt0_kernel << < grid, block >> > (camPos, camDir, camUp, camRight, fov, width, height, scene->GetCompactBVH()->nodeN, g_devResultData);

	// Copy result to host
	hipMemcpy(result, g_devResultData, g_resultDataSize, hipMemcpyDeviceToHost);
	return true;
}

void freeAllCudaMem()
{
	HANDLE_ERROR(hipUnbindTexture(g_bvhMinMaxBounds));
	HANDLE_ERROR(hipUnbindTexture(g_bvhOffsetTriStartN));
	HANDLE_ERROR(hipUnbindTexture(g_triIntersectionData));
	CUFREE(g_devTriangles);
	CUFREE(g_devMaterials);
	CUFREE(g_devResultData);
	CUFREE(g_devBVHMinMaxBounds);
	CUFREE(g_devBVHOffsetTriStartN);
	CUFREE(g_devTriIntersectionData);
}