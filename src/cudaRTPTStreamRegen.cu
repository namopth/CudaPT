#include "hip/hip_runtime.h"
#include "cudaRTCommon.h"

#include <thrust/remove.h>
#include <thrust/execution_policy.h>

#define BLOCK_SIZE 16
#define NORMALRAY_BOUND_MAX 10
#define PATHSTREAM_SIZE 1E4*64
namespace cudaRTPTStreamRegen
{
	CUDA_RT_COMMON_ATTRIBS_N(0)
	CUDA_RT_COMMON_ATTRIBS_BGN
	CUDA_RT_COMMON_ATTRIBS_END

	enum RAYTYPE
	{
		RAYTYPE_EYE = 0,
		RAYTYPE_DIFF = 1,
		RAYTYPE_SPEC = 2
	};

	struct PTPathVertex
	{
		uint isTerminated;
		uint2 pathPixel;
		float3 pathOutDir;
		float3 pathVertexPos;
		float3 pathOutMulTerm;
		RAYTYPE pathType;
		float3 pathSample;
		float3 pathAccumSample;
		uint pathSampleN;
		uint pathSampleDepth;
		hiprandState randState;

		__device__ PTPathVertex(uint _isTerminated, uint2 _pathPixel, float3 _pathOutDir, float3 _pathVertexPos, RAYTYPE _pathType, hiprandState _randState)
			: isTerminated(_isTerminated)
			, pathPixel(_pathPixel)
			, pathOutDir(_pathOutDir)
			, pathVertexPos(_pathVertexPos)
			, pathOutMulTerm(make_float3(1.f,1.f,1.f))
			, pathType(_pathType)
			, pathSample(make_float3(0.f, 0.f, 0.f))
			, pathAccumSample(make_float3(0.f, 0.f, 0.f))
			, pathSampleN(0)
			, pathSampleDepth(0)
			, randState(_randState)
		{}
	};

	PTPathVertex* g_devPathQueue = nullptr;
	uint g_uPathQueueCur = 0;
	uint g_uPathQueueSize = 0;
	PTPathVertex** g_devPathStream = nullptr;
	uint g_uPathStreamSize = PATHSTREAM_SIZE;
	PTPathVertex** g_devPathQueueRegen = nullptr;
	uint g_uPathQueueRegenSize = 0;

	void freeStreamMem()
	{
		g_uPathQueueCur = g_uPathQueueSize = 0;
		CUFREE(g_devPathQueue);
		CUFREE(g_devPathStream);
		CUFREE(g_devPathQueueRegen);
	}

	void allocateStreamMem(uint queueSize = 480000)
	{
		g_uPathQueueSize = queueSize;
		HANDLE_ERROR(hipMalloc((void**)&g_devPathQueue, sizeof(PTPathVertex) * g_uPathQueueSize));
		HANDLE_ERROR(hipMemset((void*)g_devPathQueue, 0, sizeof(PTPathVertex) * g_uPathQueueSize));

		HANDLE_ERROR(hipMalloc((void**)&g_devPathStream, sizeof(PTPathVertex*) * g_uPathStreamSize));
		HANDLE_ERROR(hipMemset((void*)g_devPathStream, 0, sizeof(PTPathVertex*) * g_uPathStreamSize));

		g_uPathQueueRegenSize = 0;
		HANDLE_ERROR(hipMalloc((void**)&g_devPathQueueRegen, sizeof(PTPathVertex*) * g_uPathQueueSize));
		HANDLE_ERROR(hipMemset((void*)g_devPathQueueRegen, 0, sizeof(PTPathVertex*) * g_uPathQueueSize));
	}

#pragma region SHADING_FUNC
	__device__ float3 Diffuse_Lambert(float3 DiffuseColor)
	{
		return DiffuseColor * (1 / M_PI);
	}

	__device__ float Vis_SmithJointApprox(float Roughness, float NoV, float NoL)
	{
		float a = Roughness * Roughness;
		float Vis_SmithV = NoL * (NoV * (1 - a) + a);
		float Vis_SmithL = NoV * (NoL * (1 - a) + a);
		return 0.5 * rcpf(Vis_SmithV + Vis_SmithL);
		//float k = (Roughness * Roughness) / 2.0f; // (Roughness + 1) * (Roughness + 1) / 8.f;
		//return (NoV / (NoV * (1 - k) + k))*(NoL / (NoL * (1 - k) + k));
	}

	__device__ float D_GGX(float Roughness, float NoH)
	{
		float m = Roughness * Roughness;
		float m2 = m*m;
		float d = (NoH * m2 - NoH) * NoH + 1;
		return m2 / (M_PI*d*d);
	}

	__device__ float3 F_Schlick(float3 SpecularColor, float VoH)
	{
		float Fc = pow(1 - VoH, 5);
		float firstTerm = saturate(50.0 * SpecularColor.z) * Fc;
		return make_float3(firstTerm, firstTerm, firstTerm) + (1 - Fc) * SpecularColor;
	}

	__device__ float3 ImportanceSampleGGX(float2 Xi, float Roughness, float3 N)
	{
		float a = Roughness * Roughness;
		float Phi = 2 * M_PI * Xi.x;
		float CosTheta = sqrt((1 - Xi.y) / (1 + (a*a - 1) * Xi.y));
		float SinTheta = sqrt(1 - CosTheta * CosTheta);
		float3 H;
		H.x = SinTheta * cos(Phi);
		H.y = SinTheta * sin(Phi);
		H.z = CosTheta;
		//float3 UpVector = abs(N.z) < 0.999 ? make_float3(0, 0, 1) : make_float3(1, 0, 0);
		//float3 TangentX = normalize(vecCross(UpVector, N));
		//float3 TangentY = vecCross(N, TangentX);

		float3 w = N;
		float3 u = normalize(vecCross((fabs(w.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), w));
		float3 v = vecCross(w, u);
		u = vecCross(v, w);

		// Tangent to world space
		return (u * H.x + v * H.y + w * H.z);
	}

	__device__ float3 Diffuse(float3 DiffuseColor, float Roughness, float NoV, float NoL, float VoH)
	{
		return Diffuse_Lambert(DiffuseColor);
	}

	__device__ float Distribution(float Roughness, float NoH)
	{
		return D_GGX(Roughness, NoH);
	}

	__device__ float GeometricVisibility(float Roughness, float NoV, float NoL, float VoH)
	{
		return Vis_SmithJointApprox(Roughness, NoV, NoL);
	}

	__device__ float3 Fresnel(float3 SpecularColor, float VoH)
	{
		return F_Schlick(SpecularColor, VoH);
	}

#pragma endregion SHADING_FUNC

	float* g_devResultData = nullptr;
	float* g_devAccResultData = nullptr;

	NPMathHelper::Mat4x4 g_matLastCamMat;
	NPMathHelper::Mat4x4 g_matCurCamMat;
	uint32 g_uCurFrameN = 0;
	size_t g_resultDataSize = 0;

	uint32 WangHash(uint32 a) {
		a = (a ^ 61) ^ (a >> 16);
		a = a + (a << 3);
		a = a ^ (a >> 4);
		a = a * 0x27d4eb2d;
		a = a ^ (a >> 15);
		return a;
	}

	__global__ void pt_traceSample_kernel(RTVertex* vertices, RTTriangle* triangles, RTMaterial* materials, CURTTexture* textures, PTPathVertex** pathStream, uint activePathStreamSize)
	{
		uint x = blockIdx.x * blockDim.x + threadIdx.x;
		if (x >= activePathStreamSize || pathStream[x]->isTerminated) return;
		PTPathVertex* procVertex = pathStream[x];
		CURay ray = CURay(procVertex->pathVertexPos, procVertex->pathOutDir);
		TracePrimitiveResult traceResult;
		if (TracePrimitive(ray, traceResult, M_INF, M_FLT_BIAS_EPSILON, false))
		{
			RTTriangle* tri = &triangles[traceResult.triId];
			RTMaterial* mat = &materials[tri->matInd];
			RTVertex* v0 = &vertices[tri->vertInd0];
			RTVertex* v1 = &vertices[tri->vertInd1];
			RTVertex* v2 = &vertices[tri->vertInd2];
			float2 uv0 = make_float2(v0->tex._x, v0->tex._y);
			float2 uv1 = make_float2(v1->tex._x, v1->tex._y);
			float2 uv2 = make_float2(v2->tex._x, v2->tex._y);
			float2 uv = uv0 * traceResult.w + uv1 * traceResult.u + uv2 * traceResult.v;
			float3 n0 = V32F3(v0->norm);
			float3 n1 = V32F3(v1->norm);
			float3 n2 = V32F3(v2->norm);
			float3 norm = n0 * traceResult.w + n1 * traceResult.u + n2 * traceResult.v;

			float3 diff;
			float3 emissive;
			float trans;
			float specular;
			float metallic;
			float roughness;
			float anisotropic;
			float sheen;
			float sheenTint;
			float clearcoat;
			float clearcoatGloss;
			GetMaterialColors(mat, uv, textures, diff, norm, emissive, trans, specular, metallic, roughness
				, anisotropic, sheen, sheenTint, clearcoat, clearcoatGloss);
			float3 nl = vecDot(norm, ray.dir) < 0.f ? norm : -1 * norm;
			{
				// Get some random microfacet
				float3 hDir = ImportanceSampleGGX(make_float2(hiprand_uniform(&procVertex->randState), hiprand_uniform(&procVertex->randState)), roughness, nl);

				// Calculate flesnel
				float voH = vecDot(-1 * ray.dir, hDir);
				float3 f0 = vecLerp(0.08 * make_float3(specular, specular, specular), diff, metallic);
				float3 brdf_f = Fresnel(f0, voH);

				// Reflected or Refracted
				float reflProb = lerp(length(brdf_f), 1.0f, metallic);
				float refrProb = trans;
				float3 reflDir;
				float3 refrDir;

				CURay nextRay = ray;
				float3 lightMulTerm;
				RAYTYPE nextRayType = procVertex->pathType;

				if (refrProb > 0)
				{
					bool into = vecDot(nl, norm) > 0.f;
					float nt = specular * 0.8f + 1.f;
					float nc = 1.0f;
					float nnt = into ? nc / nt : nt / nc;
					float ddn = vecDot(hDir, ray.dir);
					float cos2t = 1.f - nnt * nnt *(1.f - ddn * ddn);
					if (cos2t < 0.f)
					{
						refrProb = 0.f;
					}
					else
					{
						refrDir = normalize(ray.dir * nnt - hDir * (ddn*nnt + sqrtf(cos2t)));
					}
				}

				if (reflProb > 0)
				{
					reflDir = normalize(ray.dir - hDir * 2 * vecDot(hDir, ray.dir));
					if (vecDot(reflDir, nl) < 0.f)
						reflProb = 0.f;
				}

				// Reflected
				if (reflProb > 0 && hiprand_uniform(&procVertex->randState) < reflProb)
				{
					nextRay = CURay(ray.orig + (traceResult.dist - M_FLT_BIAS_EPSILON) * ray.dir, reflDir);
					// ShootRayResult nextRayResult = pt0_normalRay<depth + 1>(nextRay, vertices, triangles, materials, textures, randstate);

					// Microfacet specular = D*G*F / (4*NoL*NoV)
					// pdf = D * NoH / (4 * VoH)
					// (G * F * VoH) / (NoV * NoH)
					float VoH = vecDot(-1 * ray.dir, hDir);
					float NoV = vecDot(nl, -1 * ray.dir);
					float NoH = vecDot(nl, hDir);
					float NoL = vecDot(nl, reflDir);
					float G = GeometricVisibility(roughness, NoV, NoL, VoH);
					//shadeResult = vecMul((brdf_f * G * VoH) / (NoV * NoH * reflProb) , nextRayResult.light) + emissive;
					lightMulTerm = (brdf_f * G * VoH) / (NoV * NoH * reflProb);
					nextRayType = RAYTYPE_SPEC;
				}

				// Diffused or Transmited
				else
				{
					// Transmited
					if (refrProb > 0 && hiprand_uniform(&procVertex->randState) < refrProb)
					{
						nextRay = CURay(ray.orig + (traceResult.dist + M_FLT_BIAS_EPSILON) * ray.dir + refrDir * M_FLT_BIAS_EPSILON, refrDir);
						//ShootRayResult nextRayResult = pt0_normalRay<depth + 1>(nextRay, vertices, triangles, materials, textures, randstate);
						float cosine = vecDot(-1 * nl, refrDir);
						//shadeResult = (cosine * vecMul(diff, nextRayResult.light)) / (refrProb * (1 - reflProb)) + emissive;
						lightMulTerm = cosine * diff / (refrProb * (1 - reflProb));
						nextRayType = RAYTYPE_SPEC;
					}
					// Diffused
					else
					{
						float3 w = nl;
						float3 u = normalize(vecCross((fabs(w.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), w));
						float3 v = vecCross(w, u);
						u = vecCross(v, w);

						float r1 = 2.f * M_PI * hiprand_uniform(&procVertex->randState);
						float r2cos = sqrtf(hiprand_uniform(&procVertex->randState));
						float r2sin = 1.f - r2cos*r2cos;
						float3 diffDir = normalize(w * r2cos + u * r2sin * cosf(r1) + v * r2sin * sinf(r1));

						nextRay = CURay(ray.orig + traceResult.dist * ray.dir + diffDir * M_FLT_BIAS_EPSILON, diffDir);
						//ShootRayResult nextRayResult = pt0_normalRay<depth + 1>(nextRay, vertices, triangles, materials, textures, randstate);

						float VoH = vecDot(-1 * ray.dir, hDir);
						float NoV = vecDot(nl, -1 * ray.dir);
						float NoL = vecDot(nl, diffDir);
						//shadeResult = (M_PI * vecMul(Diffuse(diff, roughness, NoV, NoL, VoH), nextRayResult.light)) / ((1 - refrProb) * (1 - reflProb)) + emissive;
						lightMulTerm = M_PI * Diffuse(diff, roughness, NoV, NoL, VoH) / ((1 - refrProb) * (1 - reflProb));
						nextRayType = RAYTYPE_DIFF;
					}
				}

				procVertex->pathSample = procVertex->pathSample + vecMul(emissive , procVertex->pathOutMulTerm);

				float pixelContrib = length(procVertex->pathOutMulTerm) * length(lightMulTerm);

				if ((procVertex->pathType == RAYTYPE_DIFF && nextRayType == RAYTYPE_SPEC) || length(emissive) > 0.f)
					pixelContrib = 0.f;

				if (hiprand_uniform(&procVertex->randState) > pixelContrib || procVertex->pathSampleDepth + 1 >= NORMALRAY_BOUND_MAX)
				{
					procVertex->pathAccumSample = procVertex->pathAccumSample + procVertex->pathSample;
					procVertex->pathSampleN++;
					procVertex->isTerminated = true;
				}
				else
				{
					procVertex->pathOutMulTerm = vecMul(procVertex->pathOutMulTerm, lightMulTerm);
					procVertex->pathOutDir = nextRay.dir;
					procVertex->pathVertexPos = nextRay.orig;
					procVertex->pathType = nextRayType;
					procVertex->pathSampleDepth++;
				}
			}
		}
		else
		{
			procVertex->pathAccumSample = procVertex->pathAccumSample + procVertex->pathSample;
			procVertex->pathSampleN++;
			procVertex->isTerminated = true;
		}
	}

	__global__ void pt_genPathQueue_kernel(float3 camPos, float3 camDir, float3 camUp, float3 camRight, float fov,
		float width, float height, uint32 frameN, uint32 hashedFrameN, PTPathVertex* pathQueue)
	{
		uint x = blockIdx.x * blockDim.x + threadIdx.x;
		uint y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x >= width || y >= height) return;

		uint ind = (y * width + x);

		float u = (2.f * ((float)x + 0.5f) / width - 1.f) * tan(fov * 0.5f) * width / height;
		float v = (2.f * ((float)y + 0.5f) / height - 1.f) * tan(fov * 0.5f);

		hiprandState randstate;
		hiprand_init(hashedFrameN + ind, 0, 0, &randstate);
		float au = u + (hiprand_uniform(&randstate) - 0.5f) / height * tan(fov * 0.5f);
		float av = v + (hiprand_uniform(&randstate) - 0.5f) / height * tan(fov * 0.5f);

		float3 dir = normalize(camRight * au + camUp * av + camDir);

		pathQueue[ind] = PTPathVertex(false, make_uint2(x,y), dir, camPos, RAYTYPE_EYE, randstate);
	}

	__global__ void pt_assignPathStream_kernel(PTPathVertex** pathStream, uint pathStreamSize, PTPathVertex* pathQueue, uint pathQueueCur, uint pathQueueSize)
	{
		uint ind = blockIdx.x * blockDim.x + threadIdx.x;
		if (ind >= pathStreamSize)
		{
			int pathQueueInd = pathQueueCur + ind - pathStreamSize;
			PTPathVertex* assignSample = nullptr;
			if (pathQueueInd < pathQueueSize)
			{
				assignSample = &pathQueue[pathQueueInd];
			}
			pathStream[ind] = assignSample;
		}
	}

	__global__ void pt_reinitRegenPathStream_kernel(PTPathVertex** pathQueueRegen, uint pathQueueRegenReinitSize,
		float3 camPos, float3 camDir, float3 camUp, float3 camRight, float fov, float width, float height)
	{
		uint ind = blockIdx.x * blockDim.x + threadIdx.x;
		if (ind < pathQueueRegenReinitSize)
		{
			PTPathVertex* assignSample = pathQueueRegen[ind];

			float u = (2.f * ((float)assignSample->pathPixel.x + 0.5f) / width - 1.f) * tan(fov * 0.5f) * width / height;
			float v = (2.f * ((float)assignSample->pathPixel.y + 0.5f) / height - 1.f) * tan(fov * 0.5f);

			float au = u + (hiprand_uniform(&assignSample->randState) - 0.5f) / height * tan(fov * 0.5f);
			float av = v + (hiprand_uniform(&assignSample->randState) - 0.5f) / height * tan(fov * 0.5f);

			float3 dir = normalize(camRight * au + camUp * av + camDir);

			*assignSample = PTPathVertex(false, assignSample->pathPixel, dir, camPos, RAYTYPE_EYE, assignSample->randState);
		}
	}

	__global__ void pt_applyPathQueueResult_kernel(PTPathVertex* pathQueue, uint pathQueueSize, uint width, uint height, uint frameN, float* result, float* accResult)
	{
		uint x = blockIdx.x * blockDim.x + threadIdx.x;

		if (x >= pathQueueSize) return;

		// add calculating sample to the result
		if (!pathQueue[x].isTerminated)
		{
			pathQueue[x].pathAccumSample = pathQueue[x].pathAccumSample + pathQueue[x].pathSample;
			pathQueue[x].pathSampleN++;
		}

		if (pathQueue[x].pathSampleN > 0)
		{
			float3 sampleResult = pathQueue[x].pathAccumSample / (float)pathQueue[x].pathSampleN;
			float resultInf = 1.f / (float)(frameN + 1);
			float oldInf = 1.f - resultInf;
			uint ind = pathQueue[x].pathPixel.y * width + pathQueue[x].pathPixel.x;
			result[ind * 3] = max(resultInf * sampleResult.x + oldInf * result[ind * 3], 0.f);
			result[ind * 3 + 1] = max(resultInf * sampleResult.y + oldInf * result[ind * 3 + 1], 0.f);
			result[ind * 3 + 2] = max(resultInf * sampleResult.z + oldInf * result[ind * 3 + 2], 0.f);
		}
	}

	void CleanMem()
	{
		freeStreamMem();
		freeAllBVHCudaMem();
		CUFREE(g_devResultData);
		CUFREE(g_devAccResultData);
	}

	struct is_terminated
	{
		__hd__ bool operator()(const PTPathVertex* vert)
		{
			return vert->isTerminated;
		}
	};

	bool Render(NPMathHelper::Vec3 camPos, NPMathHelper::Vec3 camDir, NPMathHelper::Vec3 camUp, float fov, RTScene* scene
		, float width, float height, float* result)
	{
		// Check and allocate everything
		if (!scene || !scene->GetCompactBVH()->IsValid())
			return false;

		NPMathHelper::Vec3 camRight = camDir.cross(camUp).normalize();
		camUp = camRight.cross(camDir).normalize();

		g_matLastCamMat = g_matCurCamMat;
		g_matCurCamMat = NPMathHelper::Mat4x4::lookAt(camPos, camPos + camDir, camUp);
		g_uCurFrameN = (g_matLastCamMat != g_matCurCamMat) ? 0 : g_uCurFrameN + 1;

		if (!g_bIsCudaInit || scene->GetIsCudaDirty())
		{
			g_matLastCamMat = g_matCurCamMat;
			g_uCurFrameN = 0;
			initAllSceneCudaMem(scene);
			allocateStreamMem(width * height);

			size_t mem_tot;
			size_t mem_free;
			hipMemGetInfo(&mem_free, &mem_tot);
			std::cout << "Memory Used : " << mem_tot-mem_free << "/" << mem_tot << " -> Free " << mem_free << std::endl;
		}
		else if (scene->GetIsCudaMaterialDirty())
		{
			updateAllSceneMaterialsCudaMem(scene);
			g_uCurFrameN = 0;
		}

		if (!g_bIsCudaInit)
			return false;

		if (!g_devResultData || !g_devAccResultData || g_resultDataSize != (sizeof(float) * 3 * width * height))
		{
			g_resultDataSize = sizeof(float) * 3 * width * height;
			CUFREE(g_devResultData);
			hipMalloc((void**)&g_devResultData, g_resultDataSize);
			CUFREE(g_devAccResultData);
			hipMalloc((void**)&g_devAccResultData, g_resultDataSize);
		}

		float3 f3CamPos = V32F3(camPos);
		float3 f3CamUp = V32F3(camUp);
		float3 f3CamDir = V32F3(camDir);
		float3 f3CamRight = V32F3(camRight);

		// Kernel go here
		dim3 block1(BLOCK_SIZE*BLOCK_SIZE, 1, 1);
		dim3 block2(BLOCK_SIZE, BLOCK_SIZE, 1);
		dim3 renderGrid(ceil(width / (float)block2.x), ceil(height / (float)block2.y), 1);
		pt_genPathQueue_kernel << < renderGrid, block2 >> > (f3CamPos, f3CamDir, f3CamUp, f3CamRight, fov, width, height
			, g_uCurFrameN, WangHash(g_uCurFrameN), g_devPathQueue);
		hipDeviceSynchronize();

		uint maxActivePathStreamSize = min((uint)PATHSTREAM_SIZE, g_uPathQueueSize);
		uint activePathStreamSize = 0;
		uint activeRegenPathStreamSize = 0;
		g_uPathQueueCur = 0;
		g_uPathQueueRegenSize = 0;
		while (g_uPathQueueCur < g_uPathQueueSize || activePathStreamSize > 0)
		{
			uint tempActivePathStreamSize = activePathStreamSize;
			pt_assignPathStream_kernel << < dim3(ceil((float)PATHSTREAM_SIZE / (float)block1.x), 1, 1), block1 >> >(g_devPathStream, activePathStreamSize, g_devPathQueue, g_uPathQueueCur, g_uPathQueueSize);
			//readjust activePathStreamSize
			activePathStreamSize = min((uint)PATHSTREAM_SIZE, activePathStreamSize + (g_uPathQueueSize - g_uPathQueueCur));
			g_uPathQueueCur += activePathStreamSize - tempActivePathStreamSize;
			hipDeviceSynchronize();

			//reinit regen path
			//if (activePathStreamSize + activeRegenPathStreamSize < maxActivePathStreamSize)
			//{
			//	uint reinitPathStreamSize = min((uint)maxActivePathStreamSize - (activePathStreamSize + activeRegenPathStreamSize), g_uPathQueueRegenSize - activeRegenPathStreamSize);
			//	pt_reinitRegenPathStream_kernel << < dim3(ceil((float)reinitPathStreamSize / (float)block1.x), 1, 1), block1 >> > 
			//		(g_devPathQueueRegen, reinitPathStreamSize, f3CamPos, f3CamDir, f3CamUp, f3CamRight, fov, width, height);
			//	activeRegenPathStreamSize += reinitPathStreamSize;
			//}

			pt_traceSample_kernel << < dim3(ceil((float)activePathStreamSize / (float)block1.x), 1, 1), block1 >> > (g_devVertices, g_devTriangles, g_devMaterials, g_devTextures, g_devPathStream, activePathStreamSize);
			hipDeviceSynchronize();

			//prepare regen list
			//PTPathVertex** regenStreamEndItr = thrust::copy_if(thrust::device, g_devPathStream, g_devPathStream + activePathStreamSize, g_devPathQueueRegen + g_uPathQueueRegenSize, is_terminated());
			//g_uPathQueueRegenSize = regenStreamEndItr - g_devPathQueueRegen;

			//compact regenpathstream and find regenpathstreamsize value
			//if (activePathStreamSize < maxActivePathStreamSize)
			//{
			//	PTPathVertex** compactRegenStreamEndItr = thrust::remove_if(thrust::device, g_devPathQueueRegen, g_devPathQueueRegen + g_uPathQueueRegenSize, is_terminated());
			//	uint terminatedRegenPathStreamSize = g_uPathQueueRegenSize - (compactRegenStreamEndItr - g_devPathQueueRegen);
			//	activeRegenPathStreamSize -= terminatedRegenPathStreamSize;
			//	g_uPathQueueRegenSize -= terminatedRegenPathStreamSize;
			//}

			//compact pathstream and find activePathStreamSize value
			PTPathVertex** compactedStreamEndItr = thrust::remove_if(thrust::device, g_devPathStream, g_devPathStream + activePathStreamSize, is_terminated());
			activePathStreamSize = compactedStreamEndItr - g_devPathStream;
		}
		pt_applyPathQueueResult_kernel << < dim3(ceil((float)g_uPathQueueSize / (float)block1.x), 1, 1), block1 >> >(g_devPathQueue, g_uPathQueueSize, width, height, g_uCurFrameN, g_devResultData, g_devAccResultData);

		// Copy result to host
		hipMemcpy(result, g_devResultData, g_resultDataSize, hipMemcpyDeviceToHost);
		return true;
	}
}