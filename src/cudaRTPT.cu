#include "hip/hip_runtime.h"
#include "cudaRTCommon.h"

#define BLOCK_SIZE 16
#define NORMALRAY_BOUND_MAX 3
namespace cudaRTPT
{

	CUDA_RT_COMMON_ATTRIBS_N(0)
	CUDA_RT_COMMON_ATTRIBS_BGN
	CUDA_RT_COMMON_ATTRIBS_END

	float* g_devResultData = nullptr;
	float* g_devAccResultData = nullptr;

	NPMathHelper::Mat4x4 g_matLastCamMat;
	NPMathHelper::Mat4x4 g_matCurCamMat;
	uint32 g_uCurFrameN;
	size_t g_resultDataSize = 0;

	struct ShootRayResult
	{
		float3 light;
	};

	template <int depth = 0>
	__device__ ShootRayResult pt0_normalRay(const CURay& ray, RTVertex* vertices, RTTriangle* triangles, RTMaterial* materials, CURTTexture* textures, hiprandState *randstate)
	{
		ShootRayResult rayResult;
		if (depth > 5)
		{
			rayResult.light = make_float3(0.f, 0.f, 0.f);
			return rayResult;
		}

		TracePrimitiveResult traceResult;
		if (TracePrimitive(ray, traceResult, M_INF, M_FLT_BIAS_EPSILON, false))
		{
			RTTriangle* tri = &triangles[traceResult.triId];
			RTMaterial* mat = &materials[tri->matInd];
			RTVertex* v0 = &vertices[tri->vertInd0];
			RTVertex* v1 = &vertices[tri->vertInd1];
			RTVertex* v2 = &vertices[tri->vertInd2];
			float2 uv0 = make_float2(v0->tex._x, v0->tex._y);
			float2 uv1 = make_float2(v1->tex._x, v1->tex._y);
			float2 uv2 = make_float2(v2->tex._x, v2->tex._y);
			float2 uv = uv0 * traceResult.w + uv1 * traceResult.u + uv2 * traceResult.v;
			float3 n0 = V32F3(v0->norm);
			float3 n1 = V32F3(v1->norm);
			float3 n2 = V32F3(v2->norm);
			float3 norm = n0 * traceResult.w + n1 * traceResult.u + n2 * traceResult.v;

			float3 diff;
			float3 emissive;
			float trans;
			float specular;
			float metallic;
			float roughness;
			float anisotropic;
			float sheen;
			float sheenTint;
			float clearcoat;
			float clearcoatGloss;
			GetMaterialColors(mat, uv, textures, diff, norm, emissive, trans, specular, metallic, roughness
				, anisotropic, sheen, sheenTint, clearcoat, clearcoatGloss);
			float3 shadeResult = make_float3(0.f,0.f,0.f);
			float3 nl = vecDot(norm, ray.dir) < 0.f ? norm : -1 * norm;
#define MICROFACET_MODEL
#ifdef MICROFACET_MODEL
			{
				// Get some random microfacet
				float3 hDir = ImportanceSampleGGX(make_float2(hiprand_uniform(randstate), hiprand_uniform(randstate)), roughness, nl);

				// Calculate flesnel
				float voH = vecDot(-1 * ray.dir, hDir);
				float3 f0 = vecLerp(0.08 * make_float3(specular, specular, specular), diff, metallic);
				float3 brdf_f = Fresnel(f0, voH);

				// Reflected or Refracted
				float reflProb = lerp(length(brdf_f), 1.0f, metallic);
				float refrProb = trans;
				float3 reflDir;
				float3 refrDir;

				if (refrProb > 0)
				{
					bool into = vecDot(nl, norm) > 0.f;
					float nt = specular * 0.8f + 1.f;
					float nc = 1.0f;
					float nnt = into ? nc / nt : nt / nc;
					float ddn = vecDot(hDir, ray.dir);
					float cos2t = 1.f - nnt * nnt *(1.f - ddn * ddn);
					if (cos2t < 0.f)
					{
						refrProb = 0.f;
					}
					else
					{
						refrDir = normalize(ray.dir * nnt - hDir * (ddn*nnt + sqrtf(cos2t)));
					}
				}

				if (reflProb > 0)
				{
					reflDir = normalize(ray.dir - hDir * 2 * vecDot(hDir,ray.dir));
					if (vecDot(reflDir, nl) < 0.f)
						reflProb = 0.f;
				}

				// Reflected
				if (reflProb > 0 && hiprand_uniform(randstate) < reflProb)
				{
					CURay nextRay(ray.orig + (traceResult.dist - M_FLT_BIAS_EPSILON) * ray.dir, reflDir);
					ShootRayResult nextRayResult = pt0_normalRay<depth + 1>(nextRay, vertices, triangles, materials, textures, randstate);
					// Microfacet specular = D*G*F / (4*NoL*NoV)
					// pdf = D * NoH / (4 * VoH)
					// (G * F * VoH) / (NoV * NoH)
					float VoH = vecDot(-1 * ray.dir, hDir);
					float NoV = vecDot(nl, -1 * ray.dir);
					float NoH = vecDot(nl, hDir);
					float NoL = vecDot(nl, reflDir);
					float G = GeometricVisibility(roughness, NoV, NoL, VoH);
					shadeResult = vecMul((brdf_f * G * VoH) / (NoV * NoH * reflProb) , nextRayResult.light) + emissive;
				}

				// Diffused or Transmited
				else
				{
					// Transmited
					if (refrProb > 0 && hiprand_uniform(randstate) < refrProb)
					{
						CURay nextRay(ray.orig + (traceResult.dist + M_FLT_BIAS_EPSILON) * ray.dir + refrDir * M_FLT_BIAS_EPSILON, refrDir);
						ShootRayResult nextRayResult = pt0_normalRay<depth + 1>(nextRay, vertices, triangles, materials, textures, randstate);
						float cosine = vecDot(-1 * nl, refrDir);
						shadeResult = (cosine * vecMul(diff, nextRayResult.light)) / (refrProb * (1 - reflProb)) + emissive;
					}
					// Diffused
					else
					{
						float3 w = nl;
						float3 u = normalize(vecCross((fabs(w.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), w));
						float3 v = vecCross(w, u);
						u = vecCross(v, w);

						float r1 = 2.f * M_PI * hiprand_uniform(randstate);
						float r2cos = sqrtf(hiprand_uniform(randstate));
						float r2sin = 1.f - r2cos*r2cos;
						float3 diffDir = normalize(w * r2cos + u * r2sin * cosf(r1) + v * r2sin * sinf(r1));

						CURay nextRay(ray.orig + traceResult.dist * ray.dir + diffDir * M_FLT_BIAS_EPSILON, diffDir);
						ShootRayResult nextRayResult = pt0_normalRay<depth + 1>(nextRay, vertices, triangles, materials, textures, randstate);

						float VoH = vecDot(-1 * ray.dir, hDir);
						float NoV = vecDot(nl, -1 * ray.dir);
						float NoL = vecDot(nl, diffDir);
						shadeResult = (M_PI * vecMul(Diffuse(diff, roughness, NoV, NoL, VoH), nextRayResult.light)) / ((1 - refrProb) * (1 - reflProb)) + emissive;
					}
				}
			}
#else
			float refrProb = hiprand_uniform(randstate);
			float3 refrDir = ray.dir;
			if (refrProb < trans)
			{
				bool into = vecDot(nl, norm) > 0.f;
				float nt = specular * 0.8f + 1.f;
				float nc = 1.0f;
				float nnt = into ? nc / nt : nt / nc;
				float ddn = vecDot(nl, ray.dir);
				float cos2t = 1.f - nnt * nnt *(1.f - ddn * ddn);
				if (cos2t < 0.f)
				{
					float3 refDir = normalize(ray.dir - norm * 2 * vecDot(norm, ray.dir));
					CURay nextRay(ray.orig + traceResult.dist * ray.dir + refDir * M_FLT_BIAS_EPSILON, refDir);
					ShootRayResult nextRayResult = pt0_normalRay<depth + 1>(nextRay, vertices, triangles, materials, textures, randstate);
					float cosine = vecDot(nl, refDir);
					shadeResult = (cosine * nextRayResult.light) / trans + emissive;
				}
				else
				{
					refrDir = normalize(ray.dir * nnt - norm * ((into ? 1 : -1)*(ddn*nnt + sqrtf(cos2t))));
					float a = nt - nc;
					float b = nt + nc;
					float r0 = a * a / (b * b);
					float c = 1.f - (into ? -ddn : vecDot(refrDir, norm));
					float re = r0 + (1.f - r0)*c*c*c*c*c;
					float tr = 1.f - re;
					float p = re;
					float reflProb = hiprand_uniform(randstate);
					if (reflProb < p)
					{
						float3 refDir = normalize(ray.dir - norm * 2 * vecDot(norm, ray.dir));
						CURay nextRay(ray.orig + traceResult.dist * ray.dir + refDir * M_FLT_BIAS_EPSILON, refDir);
						ShootRayResult nextRayResult = pt0_normalRay<depth + 1>(nextRay, vertices, triangles, materials, textures, randstate);
						float cosine = vecDot(nl, refDir);
						shadeResult = (re * cosine * nextRayResult.light) / (trans * p) + emissive;
					}
					else
					{
						CURay nextRay(ray.orig + (traceResult.dist + M_FLT_BIAS_EPSILON) * ray.dir + refrDir * M_FLT_BIAS_EPSILON, refrDir);
						ShootRayResult nextRayResult = pt0_normalRay<depth + 1>(nextRay, vertices, triangles, materials, textures, randstate);
						float cosine = vecDot(-1 * nl, refrDir);
						shadeResult = (tr * cosine * nextRayResult.light) / (trans * (1.f - p)) + emissive;
					}
				}
			}
			else
			{
				float3 w = nl;
				float3 u = normalize(vecCross((fabs(w.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), w));
				float3 v = vecCross(w, u);
				u = vecCross(v, w);

				//float r1 = 2.f * M_PI * hiprand_uniform(randstate);
				//float r2 = hiprand_uniform(randstate);
				//float r2s = sqrtf(r2);
				//float3 refDir = normalize(u*cosf(r1)*r2s + v*sinf(r1)*r2s + w*sqrtf(1.f - r2));

				float r1 = 2.f * M_PI * hiprand_uniform(randstate);
				float r2 = 0.5f * M_PI * hiprand_uniform(randstate);
				float r2sin = sinf(r2);
				float3 refDir = normalize(w * cosf(r2) + u * r2sin * cosf(r1) + v * r2sin * sinf(r1));

				CURay nextRay(ray.orig + traceResult.dist * ray.dir + refDir * M_FLT_BIAS_EPSILON, refDir);
				ShootRayResult nextRayResult = pt0_normalRay<depth + 1>(nextRay, vertices, triangles, materials, textures, randstate);
				nextRayResult.light = nextRayResult.light;
				float cosine = vecDot(nl, refDir);
				shadeResult = (M_PI * cosine * vecMul(diff, nextRayResult.light)) / (1 - trans) + emissive;
			}
#endif
			rayResult.light = shadeResult;
		}
		else
		{
			rayResult.light.x = 1.f;
			rayResult.light.y = 1.f;
			rayResult.light.z = 1.f;
		}

		return rayResult;
	}

	template <>
	__device__ ShootRayResult pt0_normalRay<NORMALRAY_BOUND_MAX>(const CURay& ray, RTVertex* vertices, RTTriangle* triangles, RTMaterial* materials, CURTTexture* textures
		, hiprandState *randstate)
	{
		ShootRayResult rayResult;
		rayResult.light.x = rayResult.light.y = rayResult.light.z = 0.f;
		return rayResult;
	}

	uint32 WangHash(uint32 a) {
		a = (a ^ 61) ^ (a >> 16);
		a = a + (a << 3);
		a = a ^ (a >> 4);
		a = a * 0x27d4eb2d;
		a = a ^ (a >> 15);
		return a;
	}

	__global__ void pt0_kernel(float3 camPos, float3 camDir, float3 camUp, float3 camRight, float fov,
		float width, float height, RTVertex* vertices, RTTriangle* triangles, RTMaterial* materials, CURTTexture* textures
		, uint32 frameN, uint32 hashedFrameN, float* result, float* accResult)
	{
		uint x = blockIdx.x * blockDim.x + threadIdx.x;
		uint y = blockIdx.y * blockDim.y + threadIdx.y;
		uint ind = (y * width + x) * 3;
		int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;


		float u = (2.f * ((float)x + 0.5f) / width - 1.f) * tan(fov * 0.5f) * width / height;
		float v = (2.f * ((float)y + 0.5f) / height - 1.f) * tan(fov * 0.5f);

		hiprandState randstate;
		hiprand_init(hashedFrameN + ind, 0, 0, &randstate);
		u = u + (hiprand_uniform(&randstate) - 0.5f) / width;
		v = v + (hiprand_uniform(&randstate) - 0.5f) / height;

		float3 dir = normalize(camRight * u + camUp * v + camDir);
		CURay ray(camPos, dir);

		ShootRayResult rayResult = pt0_normalRay(ray, vertices, triangles, materials, textures, &randstate);

		float resultInf = 1.f / (float)(frameN + 1);
		float oldInf = 1.f - resultInf;
		result[ind] = max(resultInf * rayResult.light.x + oldInf * result[ind], 0.f);
		result[ind + 1] = max(resultInf * rayResult.light.y + oldInf * result[ind + 1], 0.f);
		result[ind + 2] = max(resultInf * rayResult.light.z + oldInf * result[ind + 2], 0.f);
	}

	void CleanMem()
	{
		freeAllBVHCudaMem();
		CUFREE(g_devResultData);
		CUFREE(g_devAccResultData);
	}

	bool Render(NPMathHelper::Vec3 camPos, NPMathHelper::Vec3 camDir, NPMathHelper::Vec3 camUp, float fov, RTScene* scene
		, float width, float height, float* result)
	{
		// Check and allocate everything
		if (!scene || !scene->GetCompactBVH()->IsValid())
			return false;

		NPMathHelper::Vec3 camRight = camDir.cross(camUp).normalize();
		camUp = camRight.cross(camDir).normalize();

		g_matLastCamMat = g_matCurCamMat;
		g_matCurCamMat = NPMathHelper::Mat4x4::lookAt(camPos, camPos + camDir, camUp);
		g_uCurFrameN = (g_matLastCamMat != g_matCurCamMat) ? 0 : g_uCurFrameN + 1;

		if (!g_bIsCudaInit || scene->GetIsCudaDirty())
		{
			g_matLastCamMat = g_matCurCamMat;
			g_uCurFrameN = 0;
			initAllSceneCudaMem(scene);
		}
		else if (scene->GetIsCudaMaterialDirty())
		{
			updateAllSceneMaterialsCudaMem(scene);
			g_uCurFrameN = 0;
		}

		if (!g_bIsCudaInit)
			return false;

		if (!g_devResultData || !g_devAccResultData || g_resultDataSize != (sizeof(float) * 3 * width * height))
		{
			g_resultDataSize = sizeof(float) * 3 * width * height;
			CUFREE(g_devResultData);
			hipMalloc((void**)&g_devResultData, g_resultDataSize);
			CUFREE(g_devAccResultData);
			hipMalloc((void**)&g_devAccResultData, g_resultDataSize);
		}

		float3 f3CamPos = V32F3(camPos);
		float3 f3CamUp = V32F3(camUp);
		float3 f3CamDir = V32F3(camDir);
		float3 f3CamRight = V32F3(camRight);

		// Kernel go here
		dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
		dim3 grid(width / block.x, height / block.y, 1);
		pt0_kernel << < grid, block >> > (f3CamPos, f3CamDir, f3CamUp, f3CamRight, fov, width, height, g_devVertices, g_devTriangles, g_devMaterials, g_devTextures
			, g_uCurFrameN, WangHash(g_uCurFrameN), g_devResultData, g_devAccResultData);

		// Copy result to host
		hipMemcpy(result, g_devResultData, g_resultDataSize, hipMemcpyDeviceToHost);
		return true;
	}
}